#include "hip/hip_runtime.h"
#include <string.h>
#include "easel.h"
#include "esl_dsqdata.h"
#include <x86intrin.h>
#include <math.h>
#include "esl_sse.h"
#include "hmmer.h"
#include "px_cuda.h"

#define MAX_BAND_WIDTH 2
#define NEGINFMASK 0x80808080
#define MAX(a, b, c)\
  a = (b > c) ? b:c;
//asm("max.f32 %0, %1, %2;" : "=f"(a): "f"(b), "f"(c));

//  a = (b > c) ? b:c;

#define NUM_REPS 10

char * restripe_char(char *source, int source_chars_per_vector, int dest_chars_per_vector, int source_length, int *dest_length);
int *restripe_char_to_int(char *source, int source_chars_per_vector, int dest_ints_per_vector, int source_length, int *dest_length);

__device__  uint calc_band_1(const __restrict__ uint8_t *dsq, int L, int Q, int q, uint ** rbv, volatile uint *residue_buffer){
  uint sv0 = NEGINFMASK, xE=NEGINFMASK, *rsc;
  int offset;
  // loop 1 of SSV

  rsc = rbv[*dsq] +(q <<5);
  if(threadIdx.x < 8){
    ((uint4 *)residue_buffer)[threadIdx.x] = ((uint4*)rsc)[threadIdx.x];
  }
  __syncwarp();
  #pragma unroll 4
  for(offset = (q <<5); offset < (((Q-1)<<5)); dsq++){
    offset += 32;
    rsc = rbv[*dsq] + offset;
    sv0   = __vaddss4(sv0, residue_buffer[threadIdx.x]);  
    if(threadIdx.x < 8){
      ((uint4 *)residue_buffer)[threadIdx.x] = ((uint4*)rsc)[threadIdx.x];
    }
    __syncwarp();
    xE  = __vmaxs4(xE, sv0);     
  }
    //convert step
    sv0   = __vaddss4(sv0, residue_buffer[threadIdx.x]);  
    xE  = __vmaxs4(xE, sv0);  
    sv0 = __byte_perm(sv0, __shfl_up_sync(0xffffffff, sv0, 1), 0x2107); //left-shifts sv by one byte, puts the high byte of sv_shuffle in the low byte of sv
    if(threadIdx.x == 0){
      sv0 = __byte_perm(sv0, NEGINFMASK, 0x2107); //left-shifts sv by one byte, puts the high byte of sv_shuffle in the low byte of sv
    }
  //loop 2 of SSV
  rsc = rbv[*dsq];
  if(threadIdx.x < 8){
    ((uint4 *)residue_buffer)[threadIdx.x] = ((uint4*)rsc)[threadIdx.x];
  }
  __syncwarp();
  offset = 0;
  #pragma unroll 4
  for(int row = Q-q; row < L-Q; row++){ // merge the nested loops in original SSV to support unrolling
    offset +=32;
    dsq++;
    rsc = rbv[*dsq] + offset;
    sv0   = __vaddss4(sv0, residue_buffer[threadIdx.x]);  
    if(threadIdx.x < 8){
      ((uint4 *)residue_buffer)[threadIdx.x] = ((uint4*)rsc)[threadIdx.x];
    }
    __syncwarp();
    xE  = __vmaxs4(xE, sv0);  
    if(offset == ((Q-1) <<5)){
      //convert step
      sv0   = __vaddss4(sv0, residue_buffer[threadIdx.x]);  
      xE  = __vmaxs4(xE, sv0);  
      sv0 = __byte_perm(sv0, __shfl_up_sync(0xffffffff, sv0, 1), 0x2107); //left-shifts sv by one byte, puts the high byte of sv_shuffle in the low byte of sv
      if(threadIdx.x == 0){
        sv0 = __byte_perm(sv0, NEGINFMASK, 0x2107); //left-shifts sv by one byte, puts the high byte of sv_shuffle in the low byte of sv
      }
      dsq++;
      rsc = rbv[*dsq];
      if(threadIdx.x < 8){
        ((uint4 *)residue_buffer)[threadIdx.x] = ((uint4*)rsc)[threadIdx.x];
      }
      __syncwarp();
      row++;
      offset = 0;
    }
  }

  //Loop 3 of SSV
  rsc = rbv[*dsq];
  if(threadIdx.x < 8){
    ((uint4 *)residue_buffer)[threadIdx.x] = ((uint4*)rsc)[threadIdx.x];
  }
  __syncwarp();
  #pragma unroll 4
  for(offset = 0; offset < ((Q-1) <<5);){
    offset += 32;
    dsq++;
    rsc = rbv[*dsq] + offset;

    sv0   = __vaddss4(sv0, residue_buffer[threadIdx.x]);  
    if(threadIdx.x < 8){
      ((uint4 *)residue_buffer)[threadIdx.x] = ((uint4*)rsc)[threadIdx.x];
    }
    __syncwarp(); 
    xE  = __vmaxs4(xE, sv0);
  }
  //convert step
  sv0   = __vaddss4(sv0, residue_buffer[threadIdx.x]);  
  xE  = __vmaxs4(xE, sv0);  

  sv0 = __byte_perm(sv0, __shfl_up_sync(0xffffffff, sv0, 1), 0x2107); //left-shifts sv by one byte, puts the high byte of sv_shuffle in the low byte of sv
  if(threadIdx.x == 0){
    sv0 = __byte_perm(sv0, NEGINFMASK, 0x2107); //left-shifts sv by one byte, puts the high byte of sv_shuffle in the low byte of sv
  }
  return xE;   
}

__device__  uint calc_band_2(const __restrict__ uint8_t *dsq, int L, int Q, int q, uint ** rbv, volatile uint *residue_buffer){
  uint sv0 = NEGINFMASK, sv1 = NEGINFMASK, xE=NEGINFMASK, *rsc;
  int offset;
  // loop 1 of SSV

  rsc = rbv[*dsq] + (q<<5);
  if(threadIdx.x < 16){
    ((uint4 *)residue_buffer)[threadIdx.x] = ((uint4*)rsc)[threadIdx.x];
  }
  __syncwarp();
  #pragma unroll 4

  for(offset = (q <<5); offset < (((Q-2)<<5)); dsq++){
    offset += 32;
    rsc = rbv[*dsq] + offset;
    sv0   = __vaddss4(sv0, residue_buffer[threadIdx.x]);
    sv1   = __vaddss4(sv1, residue_buffer[threadIdx.x+32]);  
    if(threadIdx.x < 16){
      ((uint4 *)residue_buffer)[threadIdx.x] = ((uint4*)rsc)[threadIdx.x];
    }
    __syncwarp();
    xE  = __vmaxs4(xE, sv0); 
    xE  = __vmaxs4(xE, sv1);     
  }
    //convert step
  sv0   = __vaddss4(sv0, residue_buffer[threadIdx.x]); 
  sv1   = __vaddss4(sv1, residue_buffer[threadIdx.x+32]);  
  xE  = __vmaxs4(xE, sv0); 
  xE  = __vmaxs4(xE, sv1);   
  sv1 = __byte_perm(sv1, __shfl_up_sync(0xffffffff, sv1, 1), 0x2107); //left-shifts sv by one byte, puts the high byte of sv_shuffle in the low byte of sv
  if(threadIdx.x == 0){
    sv1 = __byte_perm(sv1, NEGINFMASK, 0x2107); //left-shifts sv by one byte, puts the high byte of sv_shuffle in the low byte of sv
  }
  dsq++;
  offset += 32;
  rsc = rbv[*dsq]+offset;
  if(threadIdx.x < 16){
    ((uint4 *)residue_buffer)[threadIdx.x] = ((uint4*)rsc)[threadIdx.x];
  }
  __syncwarp();

  //convert step
  sv0   = __vaddss4(sv0, residue_buffer[threadIdx.x]); 
  sv1   = __vaddss4(sv1, residue_buffer[threadIdx.x+32]);  
  xE  = __vmaxs4(xE, sv0); 
  xE  = __vmaxs4(xE, sv1);   
  sv0 = __byte_perm(sv0, __shfl_up_sync(0xffffffff, sv0, 1), 0x2107); //left-shifts sv by one byte, puts the high byte of sv_shuffle in the low byte of sv
  if(threadIdx.x == 0){
    sv0 = __byte_perm(sv0, NEGINFMASK, 0x2107); //left-shifts sv by one byte, puts the high byte of sv_shuffle in the low byte of sv
  }
  dsq++;

  //loop 2 of SSV
  rsc = rbv[*dsq];
  if(threadIdx.x < 16){
    ((uint4 *)residue_buffer)[threadIdx.x] = ((uint4*)rsc)[threadIdx.x];
  }
  __syncwarp();
  offset = 0;
  #pragma unroll 4
  for(int row = Q-q; row < L-Q; row++){ // merge the nested loops in original SSV to support unrolling
    offset +=32;
    dsq++;
    rsc = rbv[*dsq] + offset;
    sv0   = __vaddss4(sv0, residue_buffer[threadIdx.x]);  
    sv1   = __vaddss4(sv1, residue_buffer[threadIdx.x+32]); 
    if(threadIdx.x < 16){
      ((uint4 *)residue_buffer)[threadIdx.x] = ((uint4*)rsc)[threadIdx.x];
    }
    __syncwarp();
    xE  = __vmaxs4(xE, sv0);  
    xE  = __vmaxs4(xE, sv1);  

    if(offset >= ((Q-2) <<5)){
      //convert step
      sv0   = __vaddss4(sv0, residue_buffer[threadIdx.x]); 
      sv1   = __vaddss4(sv1, residue_buffer[threadIdx.x+32]);  
      xE  = __vmaxs4(xE, sv0); 
      xE  = __vmaxs4(xE, sv1);   
      sv1 = __byte_perm(sv1, __shfl_up_sync(0xffffffff, sv1, 1), 0x2107); //left-shifts sv by one byte, puts the high byte of sv_shuffle in the low byte of sv
      if(threadIdx.x == 0){
        sv1 = __byte_perm(sv1, NEGINFMASK, 0x2107); //left-shifts sv by one byte, puts the high byte of sv_shuffle in the low byte of sv
      }
      dsq++;
      offset+=32;
      rsc = rbv[*dsq]+offset;
      if(threadIdx.x < 16){
        ((uint4 *)residue_buffer)[threadIdx.x] = ((uint4*)rsc)[threadIdx.x];
      }
      __syncwarp();
      row++;
      //convert step
      sv0   = __vaddss4(sv0, residue_buffer[threadIdx.x]); 
      sv1   = __vaddss4(sv1, residue_buffer[threadIdx.x+32]);  
      xE  = __vmaxs4(xE, sv0); 
      xE  = __vmaxs4(xE, sv1);   
      sv0 = __byte_perm(sv0, __shfl_up_sync(0xffffffff, sv0, 1), 0x2107); //left-shifts sv by one byte, puts the high byte of sv_shuffle in the low byte of sv
      if(threadIdx.x == 0){
        sv0 = __byte_perm(sv0, NEGINFMASK, 0x2107); //left-shifts sv by one byte, puts the high byte of sv_shuffle in the low byte of sv
      }
      dsq++;
      offset=0;
      rsc = rbv[*dsq];
      if(threadIdx.x < 16){
        ((uint4 *)residue_buffer)[threadIdx.x] = ((uint4*)rsc)[threadIdx.x];
      }
      __syncwarp();
      row++;
    }
  }

  //Loop 3 of SSV
  rsc = rbv[*dsq];
  if(threadIdx.x < 16){
    ((uint4 *)residue_buffer)[threadIdx.x] = ((uint4*)rsc)[threadIdx.x];
  }
  __syncwarp();
//  #pragma unroll 4
  for(offset = 0; offset < ((Q-2) <<5);){
    offset += 32;
    dsq++;
    rsc = rbv[*dsq] + offset;
    sv0   = __vaddss4(sv0, residue_buffer[threadIdx.x]); 
    sv1   = __vaddss4(sv1, residue_buffer[threadIdx.x+32]); 
 
    if(threadIdx.x < 16){
      ((uint4 *)residue_buffer)[threadIdx.x] = ((uint4*)rsc)[threadIdx.x];
    }
    __syncwarp(); 
    xE  = __vmaxs4(xE, sv0);
    xE  = __vmaxs4(xE, sv1);

  }
  //convert step
  sv0   = __vaddss4(sv0, residue_buffer[threadIdx.x]); 
  sv1   = __vaddss4(sv1, residue_buffer[threadIdx.x+32]);  
  xE  = __vmaxs4(xE, sv0); 
  xE  = __vmaxs4(xE, sv1);   
  sv1 = __byte_perm(sv1, __shfl_up_sync(0xffffffff, sv1, 1), 0x2107); //left-shifts sv by one byte, puts the high byte of sv_shuffle in the low byte of sv
  if(threadIdx.x == 0){
    sv1 = __byte_perm(sv1, NEGINFMASK, 0x2107); //left-shifts sv by one byte, puts the high byte of sv_shuffle in the low byte of sv
  }
  dsq++;
  offset +=32;
  rsc = rbv[*dsq]+offset;
  if(threadIdx.x < 16){
    ((uint4 *)residue_buffer)[threadIdx.x] = ((uint4*)rsc)[threadIdx.x];
  }
  __syncwarp();

  //convert step
  sv0   = __vaddss4(sv0, residue_buffer[threadIdx.x]); 
  sv1   = __vaddss4(sv1, residue_buffer[threadIdx.x+32]);  
  xE  = __vmaxs4(xE, sv0); 
  xE  = __vmaxs4(xE, sv1);   
  sv0 = __byte_perm(sv0, __shfl_up_sync(0xffffffff, sv0, 1), 0x2107); //left-shifts sv by one byte, puts the high byte of sv_shuffle in the low byte of sv
  if(threadIdx.x == 0){
    sv0 = __byte_perm(sv0, NEGINFMASK, 0x2107); //left-shifts sv by one byte, puts the high byte of sv_shuffle in the low byte of sv
  }
  return xE;   
}


__global__
void SSV_cuda(const __restrict__ uint8_t *dsq, int L, P7_OPROFILE *om, int8_t *retval){
  __shared__ uint4 shared_buffer[1024 *3];  //allocate one big lump that takes up all our shared memory
  int  Q = ((((om->M)-1) / (128)) + 1);
  int8_t *my_buffer= (int8_t*)(shared_buffer + (threadIdx.y * 96));
  uint *residue_buffer = (uint*) my_buffer;
  uint **rbv = (uint **)(my_buffer + 512); // residue buffer = 128B
  uint8_t *my_dsq_buffer = ((uint8_t *)rbv) + 224; // 27 ptrs, rounded up to multiple of 16 bytes.
  const uint8_t *dsq_ptr;
  // needs to scale w abc->Kp

  for(int i =0; i < 27; i++){
    rbv[i]=(uint *)(om->rbv[i]);
  }

  int q;
  int last_q = 0;
  uint xE = NEGINFMASK, sv_shuffle;
  int bands;
  int num_reps, i;


  for(num_reps = 0; num_reps < NUM_REPS; num_reps++){
    if(L <= 800){
      if(threadIdx.x == 0){
        memcpy(my_dsq_buffer, dsq, L);
      }
      __syncwarp();
      dsq_ptr = my_dsq_buffer;
    }
    else{
      dsq_ptr = dsq;
    }
  last_q = 0; // reset this at start of filter
  /* Use the highest number of bands but no more than MAX_BANDS */
  bands = (Q + MAX_BAND_WIDTH - 1) / MAX_BAND_WIDTH;
  for (i = 0; i < bands; i++) 
    {
      q      = (Q * (i + 1)) / bands;
      switch(q-last_q){
        case 1:
          xE = __vmaxs4(xE, calc_band_1(dsq_ptr, L, Q, last_q, rbv, residue_buffer));
          break;
       case 2:
          xE = __vmaxs4(xE, calc_band_2(dsq_ptr, L, Q, last_q, rbv, residue_buffer));
          break;

        default:
          printf("Illegal band width %d\n", q-last_q);
      }

      last_q = q;
    }
  }
// Done with main loop.  Now reduce answer vector (xE) to one byte for return
  // Reduce 32 4x8-bit quantities to 16
  sv_shuffle = __shfl_down(xE, 16); 
  if(threadIdx.x < 16){ // only bottom half of the cores continue from here
  

    xE = __vmaxs4(xE, sv_shuffle);

    // Reduce 6 4x8-bit quantities to 8
    sv_shuffle = __shfl_down(xE, 8); 
    if(threadIdx.x < 8){ // only bottom half of the cores continue from here
  

      xE = __vmaxs4(xE, sv_shuffle);

      // Reduce 8 4x8-bit quantities to 4
      sv_shuffle = __shfl_down(xE, 4); 
      if(threadIdx.x < 4){ // only bottom half of the cores continue from here

        xE = __vmaxs4(xE, sv_shuffle);

        // Reduce 4 4x8-bit quantities to 2
        sv_shuffle = __shfl_down(xE, 2); 
        if(threadIdx.x < 2){ // only bottom half of the cores continue from here

          xE = __vmaxs4(xE, sv_shuffle);
          // Reduce 2 4x8-bit quantities to 1

          sv_shuffle = __shfl_down(xE, 1);  
          if(threadIdx.x < 1){ // only bottom half of the cores continue from here

            xE = __vmaxs4(xE, sv_shuffle);

            // now, reduce the final 32 bit quantity to one 8-bit quantity.

            sv_shuffle = xE >> 16;

            xE = __vmaxs4(xE, sv_shuffle);

            sv_shuffle = xE >> 8;

            xE = __vmaxs4(xE, sv_shuffle);
            if((blockIdx.y == 0) &&(threadIdx.y ==0) && (threadIdx.x == 0)){ // only one thread writes result
              *retval = xE & 255; // low 8 bits of the word is the final result
            }
          }
        }
      }
    }
  }

  return; 
}  

/*
__global__ 
//__launch_bounds__(1024,2)
void SSV_cuda_takeone(const __restrict__ uint8_t *dsq, int L, const __restrict__ P7_OPROFILE *om, int8_t *retval){
  __shared__ uint shared_buffer[1024 *12];  //allocate one big lump that takes up all our shared memory
  int8_t *my_buffer; 
  my_buffer = (int8_t *)(shared_buffer + (threadIdx.y * 384));
  int8_t *dsq_ptr;
  int Q;
  uint sv_shuffle, *rsc;
  Q = ((((om->M)-1) / (128)) + 1);
  int xE, row, column, sv, sv0, sv1, sv2, sv3;
  uint **rbv = (uint **)my_buffer + (L+(sizeof(uint)-1))/sizeof(uint);
  for(int i =0; i < 27; i++){
    rbv[i]=(uint *)om->rbv[i];
  }
  for(int reps = 0; reps <NUM_REPS; reps++){
    if(threadIdx.x ==0){
      memcpy(my_buffer, (void *)dsq, L);
    }
    xE = NEGINFMASK; 

    for (int band = 0; band < Q/4; band+=4) {
      if(threadIdx.x==0){
              printf("Running band %d, Q/4 = %d\n", band, Q/4);
            }
        sv0 = NEGINFMASK;
        sv1 = NEGINFMASK;
        sv2 = NEGINFMASK;
        sv3 = NEGINFMASK;
        dsq_ptr = my_buffer;
        int offset;
        // loop 1 of SSV
        for(row = 0, offset = (band <<5)+threadIdx.x; row < (Q-band-4); row++){
            rsc = rbv[*dsq_ptr] + offset;
            offset += 128;
            dsq_ptr++;
            sv0   = __vaddss4(sv0, *rsc);
            sv1   = __vaddss4(sv1, *(rsc+32));
            sv2   = __vaddss4(sv2, *(rsc+64));
            sv3   = __vaddss4(sv3, *(rsc+96));
            xE  = __vmaxs4(xE, sv0); 
            xE  = __vmaxs4(xE, sv1); 
            xE  = __vmaxs4(xE, sv2);
            xE  = __vmaxs4(xE, sv3);      
        }
        //convert step
        rsc = rbv[*dsq_ptr] + offset;
        sv0   = __vaddss4(sv0, *rsc); 
        xE  = __vmaxs4(xE, sv);  
        sv_shuffle = __shfl_up(sv0, 1);  // Get the next core up's value of SV
        if(threadIdx.x == 0){
          sv0 = __byte_perm(sv0, NEGINFMASK, 0x2107); //left-shifts sv by one byte, puts the high byte of neginfmask in the low byte of sv
        }
        else{
          sv0 = __byte_perm(sv0, sv_shuffle, 0x2107); //left-shifts sv by one byte, puts the high byte of sv_shuffle in the low byte of sv
        }
        //loop 2 of SSV
        for(; row < L-Q; row++, dsq_ptr++){
          rsc = rbv[*dsq_ptr] + threadIdx.x;
          for(offset =threadIdx.x, column =0; column < Q-1; column++, row++){
            sv0   = __vaddss4(sv0, *rsc);
            sv1 =  __vaddss4(sv1, *(rsc+32));
            sv2 =  __vaddss4(sv2, *(rsc+64));
            sv3 =  __vaddss4(sv3, *(rsc+96));
            xE  = __vmaxs4(xE, sv0);  
            xE  = __vmaxs4(xE, sv1);  
            xE  = __vmaxs4(xE, sv2);  
            xE  = __vmaxs4(xE, sv3);   
            dsq_ptr++;
            offset +=128;
            rsc = rbv[*dsq_ptr] + offset;
            xE  = __vmaxs4(xE, sv);  
          }
           //convert step
          rsc = rbv[*dsq_ptr] + offset;
          sv0   = __vaddss4(sv0, *rsc); 
          xE  = __vmaxs4(xE, sv0);  

          sv_shuffle = __shfl_up(sv0, 1);  // Get the next core up's value of SV
          if(threadIdx.x == 0){
            sv0 = __byte_perm(sv0, NEGINFMASK, 0x2107); //left-shifts sv by one byte, puts the high byte of neginfmask in the low byte of sv
          }
          else{
            sv0 = __byte_perm(sv0, sv_shuffle, 0x2107); //left-shifts sv by one byte, puts the high byte of sv_shuffle in the low byte of sv
          }
        }

        //Loop 3 of SSV
        for(column = 0, offset = threadIdx.x; column < (Q-1); column++){
          rsc = rbv[my_buffer[row]] + offset;
          row++; 
          offset += 32;
          sv0   = __vaddss4(sv0, *rsc); 
          xE  = __vmaxs4(xE, sv0);
        }
        //convert step
        rsc = rbv[my_buffer[row]] + offset;
        sv0   = __vaddss4(sv0, *rsc); 
        xE  = __vmaxs4(xE, sv0);  
        sv_shuffle = __shfl_up(sv0, 1);  // Get the next core up's value of SV
        if(threadIdx.x == 0){
          sv0 = __byte_perm(sv0, NEGINFMASK, 0x2107); //left-shifts sv by one byte, puts the high byte of neginfmask in the low byte of sv
        }
        else{
          sv0 = __byte_perm(sv0, sv_shuffle, 0x2107); //left-shifts sv by one byte, puts the high byte of sv_shuffle in the low byte of sv
        }   
    }
  }

  // Done with main loop.  Now reduce answer vector (xE) to one byte for return
  // Reduce 32 4x8-bit quantities to 16
  sv_shuffle = __shfl_down(xE, 16); 
  if(threadIdx.x < 16){ // only bottom half of the cores continue from here
  

    xE = __vmaxs4(xE, sv_shuffle);

    // Reduce 6 4x8-bit quantities to 8
    sv_shuffle = __shfl_down(xE, 8); 
    if(threadIdx.x < 8){ // only bottom half of the cores continue from here
  

      xE = __vmaxs4(xE, sv_shuffle);

      // Reduce 8 4x8-bit quantities to 4
      sv_shuffle = __shfl_down(xE, 4); 
      if(threadIdx.x < 4){ // only bottom half of the cores continue from here

        xE = __vmaxs4(xE, sv_shuffle);

        // Reduce 4 4x8-bit quantities to 2
        sv_shuffle = __shfl_down(xE, 2); 
        if(threadIdx.x < 2){ // only bottom half of the cores continue from here

          xE = __vmaxs4(xE, sv_shuffle);
          // Reduce 2 4x8-bit quantities to 1

          sv_shuffle = __shfl_down(xE, 1);  
          if(threadIdx.x < 1){ // only bottom half of the cores continue from here

            xE = __vmaxs4(xE, sv_shuffle);

            // now, reduce the final 32 bit quantity to one 8-bit quantity.

            sv = xE >> 16;

            xE = __vmaxs4(xE, sv);

            sv = xE >> 8;

            xE = __vmaxs4(xE, sv);
            if((blockIdx.y == 0) &&(threadIdx.y ==0) && (threadIdx.x == 0)){ // only one thread writes result
              *retval = xE & 255; // low 8 bits of the word is the final result
            }
          }
        }
      }
    }
  }

  return;
}  
*/

// GPU kernel that copies values from the CPU version of an OPROFILE to one on the GPU.  Should generally only be called on one GPU core
__global__ void copy_oprofile_values_to_card(P7_OPROFILE *the_profile, float tauBM, float scale_b, float scale_w, int16_t base_w, int16_t ddbound_w, int L, int M, int V, int max_length, int allocM, int allocQb, int allocQw, int allocQf, int mode, float nj, int is_shadow, int8_t **rbv){

  the_profile->tauBM = tauBM;
  the_profile->scale_b = scale_b;
  the_profile->scale_w = scale_w;
  the_profile->base_w = base_w;
  the_profile->ddbound_w = ddbound_w;
  the_profile->L = L;
  the_profile->M = M;
  the_profile->V = V;
  the_profile->max_length = max_length;
  the_profile->allocM = allocM;
  the_profile->allocQb = allocQb;
  the_profile->allocQw = allocQw;
  the_profile->allocQf = allocQf;
  the_profile->mode = mode;
  the_profile->nj = nj;
  the_profile->is_shadow = is_shadow;
  the_profile->rbv = rbv;
}


// GPU kernel that initializes a filtermx structure
__global__ void initialize_filtermx_on_card(P7_FILTERMX *the_filtermx){
  the_filtermx->M = 0;
  the_filtermx->Vw = 64; // 32 cores * 32 bits = 1024 bits = 128 bytes = 64 * 16 bits
  the_filtermx->allocM = 0;
  the_filtermx->dp = NULL;
  the_filtermx->type = p7F_SSVFILTER;
}


// allocates and populates a P7_OPROFILE structure on a CUDA card that matches the one passed as its argument
P7_OPROFILE *create_oprofile_on_card(P7_OPROFILE *the_profile){
  P7_OPROFILE *cuda_OPROFILE;
  hipError_t err;
  int Q = P7_Q(the_profile->M, the_profile->V);

  if(hipMalloc(&cuda_OPROFILE, sizeof(P7_OPROFILE)) != hipSuccess){

    err = hipGetLastError();
    printf("Error: %s\n", hipGetErrorString(err));
    p7_Fail((char *) "Unable to allocate memory in create_oprofile_on_card");
  }

  // allocate and copy over rbv 2-D array
  unsigned int **cuda_rbv;
  if(hipMalloc(&cuda_rbv, the_profile->abc->Kp * sizeof(unsigned int *)) != hipSuccess){
    p7_Fail((char *) "Unable to allocate memory in create_oprofile_on_card");
  }
  int i;
  char *restriped_rbv;
  int restriped_rbv_size;

  unsigned int **cuda_rbv_temp = cuda_rbv; // use this variable to copy rbv pointers into CUDA array 
  for(i = 0; i < the_profile->abc->Kp; i++){
    int *cuda_rbv_entry;
  restriped_rbv = restripe_char ((char*)(the_profile->rbv[i]), the_profile->V, 128, Q * the_profile->V, &restriped_rbv_size);
  //restriped_rbv = (int *) restripe_char((char *)(the_profile->rbv[i]), the_profile->V, 128, Q * the_profile->V, &restriped_rbv_size);

    if(hipMalloc(&cuda_rbv_entry, restriped_rbv_size) != hipSuccess){
      p7_Fail((char *) "Unable to allocate memory in create_oprofile_on_card");
    }

    if(hipMemcpy(cuda_rbv_entry, restriped_rbv, restriped_rbv_size, hipMemcpyHostToDevice) != hipSuccess){
      p7_Fail((char *) "Unable to copy data in create_oprofile_on_card");
    }

    if(hipMemcpy(cuda_rbv_temp, &cuda_rbv_entry, sizeof(int *) , hipMemcpyHostToDevice) != hipSuccess){
      p7_Fail((char *) "Unable to copy data in create_oprofile_on_card");
    }
    cuda_rbv_temp +=1;
  }
 

  // copy over base parameters.  Only call this kernel on one core because it just assigns values to fields in the data structure and has no parallelism
  copy_oprofile_values_to_card<<<1,1>>>(cuda_OPROFILE, the_profile->tauBM, the_profile->scale_b, the_profile->scale_w, the_profile->base_w, the_profile->ddbound_w, the_profile->L, the_profile->M, the_profile->V, the_profile->max_length, the_profile->allocM, the_profile->allocQb, the_profile->allocQw, the_profile->allocQf, the_profile->mode, the_profile->nj, the_profile->is_shadow, (int8_t **) cuda_rbv);

 return cuda_OPROFILE;
}

void destroy_oprofile_on_card(P7_OPROFILE *cpu_oprofile, P7_OPROFILE *cuda_oprofile){
  int i;
  for(i = 0; i < cpu_oprofile->abc->Kp; i++){
    hipFree(cuda_oprofile->rbv[i]);
  }
  hipFree(cuda_oprofile->rbv);
  hipFree(cuda_oprofile);
}

P7_FILTERMX *create_filtermx_on_card(){
  P7_FILTERMX *the_filtermx;
  
  if(hipMalloc(&the_filtermx, sizeof(P7_FILTERMX)) != hipSuccess){
    p7_Fail((char *) "Unable to allocate memory in create_filtermx_on_card");
  }
  initialize_filtermx_on_card<<<1,1>>>(the_filtermx);
  return the_filtermx;
}


char * restripe_char(char *source, int source_chars_per_vector, int dest_chars_per_vector, int source_length, int *dest_length){
  char *dest;
  int dest_num_vectors, unpadded_dest_vectors;
  int source_num_vectors;
  source_num_vectors = source_length/source_chars_per_vector;
  if(source_num_vectors * source_chars_per_vector != source_length){
    source_num_vectors++;
  }
  unpadded_dest_vectors = source_length/dest_chars_per_vector;
  if(unpadded_dest_vectors * dest_chars_per_vector != source_length){
    unpadded_dest_vectors++;  //round up if source length isn't a multiple of the dest vector length
  }
 // printf("Unpadded_dest_vectors = %d. ", unpadded_dest_vectors);
  dest_num_vectors = unpadded_dest_vectors + MAX_BAND_WIDTH -1; // add extra vectors for SSV wrap-around

  dest = (char *) malloc(dest_num_vectors * dest_chars_per_vector);
  *dest_length = dest_num_vectors * dest_chars_per_vector;

  int source_pos, dest_pos;
  int i;

  for(i = 0; i < source_length; i++){
    source_pos = ((i % source_num_vectors) * source_chars_per_vector) + (i / source_num_vectors);
    dest_pos = ((i % unpadded_dest_vectors) * dest_chars_per_vector) + (i / unpadded_dest_vectors);
    dest[dest_pos] = (int) source[source_pos];
  }

  // pad out the dest vector with zeroes if necessary
  for(; i < unpadded_dest_vectors * dest_chars_per_vector; i++){
      dest_pos = ((i % unpadded_dest_vectors) * dest_chars_per_vector) + (i / unpadded_dest_vectors);
    //  printf("Padding 0 at location %d \n", dest_pos);
    dest[dest_pos] = 0;
  }

  // add the extra copies of the early vectors to support the SSV wrap-around
  for(int source = 0; i < dest_num_vectors * dest_chars_per_vector; i++){
    dest[i] = dest[source];
   // printf("Padding from location %d to location %d\n", source, i);
    source++;
  }

  return dest;

}


/*int *restripe_char_to_int(char *source, int source_chars_per_vector, int dest_ints_per_vector, int source_length, int *dest_length){
  int *dest;
  int dest_num_vectors, source_num_vectors, unpadded_dest_vectors;

  source_num_vectors = source_length/source_chars_per_vector;
  if(source_num_vectors * source_chars_per_vector != source_length){
    source_num_vectors++;
  }
  unpadded_dest_vectors = source_length/dest_ints_per_vector;
  if(unpadded_dest_vectors * dest_ints_per_vector != source_length){
    unpadded_dest_vectors++;  //round up if source length isn't a multiple of the dest vector length
  }
 // printf("Unpadded_dest_vectors = %d. ", unpadded_dest_vectors);
  dest_num_vectors = unpadded_dest_vectors + MAX_BAND_WIDTH -1; // add extra vectors for SSV wrap-around
  dest = (int *) malloc(dest_num_vectors * dest_ints_per_vector * sizeof(int));
  *dest_length = dest_num_vectors * dest_ints_per_vector *sizeof(int);
  //printf("Padded dest_num_vectors = %d. Dest_length = %d\n", dest_num_vectors, *dest_length);

  int source_pos, dest_pos;
  int i;

  for(i = 0; i < source_length; i++){
    source_pos = ((i % source_num_vectors) * source_chars_per_vector) + (i / source_num_vectors);
    dest_pos = ((i % unpadded_dest_vectors) * dest_ints_per_vector) + (i / unpadded_dest_vectors);
    dest[dest_pos] = (int) source[source_pos];
  }

  // pad out the dest vector with zeroes if necessary
  for(; i < unpadded_dest_vectors * dest_ints_per_vector; i++){
      dest_pos = ((i % unpadded_dest_vectors) * dest_ints_per_vector) + (i / unpadded_dest_vectors);
    //  printf("Padding 0 at location %d \n", dest_pos);
    dest[dest_pos] = 0;
  }

  // add the extra copies of the early vectors to support the SSV wrap-around
  for(int source = 0; i < dest_num_vectors * dest_ints_per_vector; i++){
    dest[i] = dest[source];
   // printf("Padding from location %d to location %d\n", source, i);
    source++;
  }

  return dest;
}
*/

int
p7_SSVFilter_shell_sse(const ESL_DSQ *dsq, int L, const __restrict__
  P7_OPROFILE *om, P7_FILTERMX *fx, float *ret_sc, P7_OPROFILE *card_OPROFILE, P7_FILTERMX *card_FILTERMX)
{
  int      Q          = P7_Q(om->M, p7_VWIDTH_SSE);
  __m128i  hv         = _mm_set1_epi8(-128);
  __m128i  neginfmask = _mm_insert_epi8( _mm_setzero_si128(), -128, 0);
  __m128i *dp;
  __m128i *rbv;
  __m128i  mpv;
  __m128i  sv;
  int8_t   h, *card_h;
  int      i,q;
  int      status;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds, seconds, gcups;
  //char *card_rbv= NULL;
  uint8_t *card_dsq;
  int warps_per_block;
  dim3 threads_per_block, num_blocks;
  hipError_t err;
  if (( status = p7_filtermx_Reinit(fx, om->M) ) != eslOK) goto FAILURE;
  fx->M    = om->M;
  fx->Vw   = p7_VWIDTH_SSE / sizeof(int16_t); // A hack. FILTERMX wants Vw in units of int16_t. 
  fx->type = p7F_SSVFILTER;
  dp       = (__m128i *) fx->dp;
  hipMalloc((void **) &card_h, 1);
  err = hipGetLastError();
  hipMalloc((void**)  &card_dsq, L+8);  //Pad out so that we can grab dsq four bytes at a time
  hipMemcpy(card_dsq, (dsq+ 1), L, hipMemcpyHostToDevice);

 // SSV_start_cuda<<<1, 32>>>(card_FILTERMX, (unsigned int *) card_hv, (unsigned int *) card_mpv, om->M);
  hipEventRecord(start);
  num_blocks.x = 1;
  num_blocks.y = 20;
  num_blocks.z = 1;
  warps_per_block = 32;
  threads_per_block.x = 32;
  threads_per_block.y = warps_per_block;
  threads_per_block.z = 1;
  /*
  int **check_array, **check_array_cuda;
  check_array = (int **) malloc(L * sizeof(int *));
  hipMalloc((void **) &check_array_cuda, (L * sizeof(int *)));
  for(int temp = 0; temp < L; temp++){
    int *row_array;
    hipMalloc((void **) &row_array, (om->M * sizeof(int)));
    check_array[temp] = row_array;
  }
  hipMemcpy(check_array_cuda, check_array, (L* sizeof(int)), hipMemcpyHostToDevice);
 */

  SSV_cuda <<<num_blocks, threads_per_block>>>(card_dsq, L, card_OPROFILE, card_h);
  int8_t h_compare;
  hipMemcpy(&h_compare, card_h, 1, hipMemcpyDeviceToHost);
  hipEventRecord(stop);

  hipEventSynchronize(stop);
  milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  seconds = milliseconds/1000;
  gcups = ((((float) (om->M * L) *(float) NUM_REPS)/seconds)/1e9) * (float)(num_blocks.x * num_blocks.y *num_blocks.z) * (float)warps_per_block;
  printf("M = %d, L = %d, seconds = %f, GCUPS = %f\n", om->M, L, seconds, gcups); 
  

  err = hipGetLastError();
  if(err != hipSuccess){
    printf("Error: %s\n", hipGetErrorString(err));
  }
 
  mpv = hv;
  for (q = 0; q < Q; q++)
    dp[q] = hv;

  for (i = 1; i <= L; i++)
    {
      rbv = (__m128i *) om->rbv[dsq[i]];

      
      for (q = 0; q < Q; q++)
        {
          sv    = _mm_adds_epi8(mpv, rbv[q]);
          hv    = _mm_max_epi8(hv, sv);
          mpv   = dp[q];
          dp[q] = sv;
        }  
      mpv = esl_sse_rightshift_int8(sv, neginfmask);

      // Compare CUDA result and SSE
 /*     hipMemcpy(restriped_rbv, card_dp, card_rbv_length, hipMemcpyDeviceToHost);
      for(j = 0; j < om->M; j++){
          int cuda_index = (j/card_Q) + ((j %card_Q) * 128);
          int cpu_index = (j/Q) + ((j %Q) * om->V);
          char *cuda_dp_char = (char *)restriped_rbv;
          char *cpu_dp_char = (char *) dp;
          char cuda_value = cuda_dp_char[cuda_index];
          char cpu_value = cpu_dp_char[cpu_index];
          if(cpu_value != cuda_value){
            printf("SSV dp miss-match at row %d, position %d: %d (CUDA) vs %d (CPU), indices were %d (CUDA), %d (CPU)\n", i, j, cuda_value, cpu_value, cuda_index, cpu_index);
          }
      }
*/
    //  free(restriped_rbv);
    }
  h = esl_sse_hmax_epi8(hv);
  hipFree(card_h);
  hipFree(card_dsq);
// hipFree(check_array_cuda)

  if(h != h_compare){
    printf("Final result miss-match: %d (CUDA) vs %d (CPU)\n", h_compare, h);
  }

  if (h == 127)  
    { *ret_sc = eslINFINITY; return eslERANGE; }
  else if (h > -128)
    { 
      *ret_sc = ((float) h + 128.) / om->scale_b + om->tauBM - 2.0;   // 2.0 is the tauNN/tauCC "2 nat approximation"
      *ret_sc += 2.0 * logf(2.0 / (float) (L + 2));                   
      return eslOK;
    }
  else 
    {
      *ret_sc = -eslINFINITY;
      return eslOK;
    }
    
 FAILURE:
  *ret_sc = -eslINFINITY;
  return status;
}


static ESL_OPTIONS options[] = {
  /* name           type      default  env  range  toggles reqs incomp  help                               docgroup*/
  { (char *) "-h",        eslARG_NONE,  FALSE,  NULL, NULL,   NULL,  NULL, NULL,  (char *) "show brief help on version and usage",  0 },
  {  (char *) "-s",        eslARG_INT,      (char *) "0",  NULL, NULL,   NULL,  NULL, NULL,  (char *) "set random number seed to <n>",         0 },
  {  0, 0, 0, 0, 0, 0, 0, 0, 0, 0 },
};
static char usage[]  = "[-options] <hmmfile> <seqfile>";
static char banner[] = "px, the first parallel tests of H4";

int
main(int argc, char **argv)
{
  ESL_GETOPTS    *go      = p7_CreateDefaultApp(options, 2, argc, argv, banner, usage);
  char           *hmmfile = esl_opt_GetArg(go, 1);
  char           *seqfile = esl_opt_GetArg(go, 2);
  ESL_ALPHABET   *abc     = NULL;
  P7_HMMFILE     *hfp     = NULL;
  P7_BG          *bg      = NULL;
  P7_HMM         *hmm     = NULL;
  P7_PROFILE     *gm      = NULL;
  P7_OPROFILE    *om      = NULL;
  ESL_DSQDATA    *dd      = NULL;
  P7_ENGINE      *eng     = NULL;
  ESL_DSQDATA_CHUNK *chu = NULL;
  int             ncore   = 1;
  int  i;
  int             status;

  /* Read in one HMM */
  if (p7_hmmfile_OpenE(hmmfile, NULL, &hfp, NULL) != eslOK) p7_Fail( (char *) "Failed to open HMM file %s", hmmfile);
  if (p7_hmmfile_Read(hfp, &abc, &hmm)            != eslOK) p7_Fail( (char *) "Failed to read HMM");
  
  /* Configure a profile from the HMM */
  bg = p7_bg_Create(abc);
  gm = p7_profile_Create (hmm->M, abc);
  om = p7_oprofile_Create(hmm->M, abc);
  p7_profile_Config   (gm, hmm, bg);
  p7_oprofile_Convert (gm, om);
  P7_OPROFILE *card_OPROFILE;
  card_OPROFILE = create_oprofile_on_card((P7_OPROFILE *) om);
  P7_FILTERMX *card_FILTERMX;
  card_FILTERMX = create_filtermx_on_card();
  p7_bg_SetFilter(bg, om->M, om->compo);

  //uint64_t sequence_id = 0;
  uint64_t num_hits = 0;
  int count;
  hipGetDeviceCount(&count);
  printf("Found %d CUDA devices\n", count);
  /* Open sequence database */
  status = esl_dsqdata_Open(&abc, seqfile, ncore, &dd);
  if      (status == eslENOTFOUND) p7_Fail( (char *) "Failed to open dsqdata files:\n  %s",    dd->errbuf);
  else if (status == eslEFORMAT)   p7_Fail( (char *) "Format problem in dsqdata files:\n  %s", dd->errbuf);
  else if (status != eslOK)        p7_Fail( (char *) "Unexpected error in opening dsqdata (code %d)", status);

  eng = p7_engine_Create(abc, NULL, NULL, gm->M, 400);

  while (( status = esl_dsqdata_Read(dd, &chu)) == eslOK && num_hits < 5)  
    {
      for (i = 0; i < 5 /* chu->N */; i++)
	{
	  p7_bg_SetLength(bg, (int) chu->L[i]);            // TODO: remove need for cast
	  p7_oprofile_ReconfigLength(om, (int) chu->L[i]); //         (ditto)
	  
	  //	  printf("seq %d %s\n", chu->i0+i, chu->name[i]);
    float ssv_score;

    p7_SSVFilter_shell_sse(chu->dsq[i], chu->L[i], om, eng->fx ,&ssv_score, card_OPROFILE, card_FILTERMX);
	 

	  p7_engine_Reuse(eng);
    num_hits++;
	}
      esl_dsqdata_Recycle(dd, chu);
    }
    printf("Saw %ld sequences\n", num_hits);
  /*esl_dsqdata_Close(dd);
  p7_oprofile_Destroy(om);
  p7_profile_Destroy(gm);
  p7_hmm_Destroy(hmm);
  p7_bg_Destroy(bg);
  p7_hmmfile_Close(hfp);
  esl_alphabet_Destroy(abc);
  esl_getopts_Destroy(go); */
  //destroy_oprofile_on_card(om, card_OPROFILE);
  exit(0);
}





