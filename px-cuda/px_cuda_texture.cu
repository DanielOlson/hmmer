#include "hip/hip_runtime.h"
#include <string.h>
#include "easel.h"
#include "esl_dsqdata.h"
#include <x86intrin.h>
#include <math.h>
#include "esl_sse.h"
#include "hmmer.h"
#include "px_cuda.h"
//#define TRUNCATE 
#define MAX(a, b, c)\
  asm("max.u32 %0, %1, %2;" : "=r"(a): "r"(b), "r"(c));



#define NUM_REPS 1
#define MAX_BAND_WIDTH 3 /*  Maximum number of registers to use in holding a "band" of the array.
This is the equivalent quantity to the orignal SSV filter's MAX_BANDS variable, just renamed in a way
I find less confusing.  Performance will probably be optimized if this is set to the largest value that
keeps the number of registers the SSV filter uses to 32 or fewer.  (64K registers per SM/32 registers per
thread allows 2048 threads/SM, which is the max)
 */
char * restripe_char(char *source, int source_chars_per_vector, int dest_chars_per_vector, int source_length, int *dest_length);
int *restripe_char_to_int(char *source, int source_chars_per_vector, int dest_ints_per_vector, int source_length, int *dest_length);

__shared__ int dsq_buffer[32];
__shared__ int8_t rbv_block[21*2048];
__shared__ int8_t *rbv_shared[21];

// Perform the core calculation on one vector
#define STEP_SINGLE(sv) \
  sv = sv + *rsc; \
  rsc += 32; \
  if (sv < -128){  \
    sv = -128;  \
  } \
  if (sv > 127){ \
    sv = 127; \
  } \
  xEv = max(xEv, sv); 


#define LENGTH_CHECK(label) \
  if (i >= L) goto label;

#define NO_CHECK(label)

#define STEP_BANDS_1() \
  STEP_SINGLE(sv[0])

#define STEP_BANDS_2() \
  STEP_BANDS_1()       \
  STEP_SINGLE(sv[1])

#define STEP_BANDS_3() \
  STEP_BANDS_2()       \
  STEP_SINGLE(sv[2])

// Here we left-shift by 5 rather than multiplying by 32 (# of threads in a warp)
// because CUDA seems to treat something as a char and generate negative indices if we multiply
// by the constant 32
#define CONVERT_STEP(step, length_check, label, sv, pos) \
  length_check(label)                                    \
  rsc = rbv_base[dsq[i]] + (pos << 5) + myoffset;               \
  if((*rsc > 127) || (*rsc < -128)){ \
    printf("Out of bounds rsc of %d found at row %d, position %d. DSQ was %d, myoffset was %d\n", *rsc, i, pos, dsq[i], myoffset);  \
  } \
  step()                                                 \
  sv_shuffle = __shfl_up(sv, 1);   \
  if(myoffset == 0){   \
    sv = neginfmask; \
  } \
  else{ \
    sv = sv_shuffle;  \
  }  \
  i++;


#define CONVERT_1(step, LENGTH_CHECK, label)            \
  CONVERT_STEP(step, LENGTH_CHECK, label, sv[0], Q - 1)

#define CONVERT_2(step, LENGTH_CHECK, label)            \
  CONVERT_STEP(step, LENGTH_CHECK, label, sv[1], Q - 2)  \
  CONVERT_1(step, LENGTH_CHECK, label)

#define CONVERT_3(step, LENGTH_CHECK, label)            \
  CONVERT_STEP(step, LENGTH_CHECK, label, sv[2], Q - 3)  \
  CONVERT_2(step, LENGTH_CHECK, label)

// declare sv as an array because CUDA can map arrays to registers and this makes it
// easier to tune the number of registers we use

// again, left-shift by 5 rather than multiplying by 32 to avoid weird negative number results
#define RESET_1()                  \
  int sv[MAX_BAND_WIDTH];          \
  sv[0] = neginfmask;

#define RESET_2() \
  RESET_1() \
  sv[1] = neginfmask;

#define RESET_3()                  \
  RESET_2()                        \
  sv[2] = neginfmask;

#define CALC(reset, step, convert, width, check_array) \
  int myoffset = threadIdx.x; \
  int neginfmask = -128; \
  int i, i2, *rsc, num_iters, sv_shuffle; \
  dsq++; \
  reset(); \
  if (L <= Q-q-width)  num_iters = L;               \
  else           num_iters = Q-q-width;           \
  i = 0;                                        \
  while (num_iters >0) {                        \
    rsc = rbv_base[dsq[i]] + ((i + q) << 5) + myoffset;  \
    step()                                      \
    i++;                                        \
    num_iters--;                                \
  }                                             \
  i = Q - q - width;                                \
  convert(step, LENGTH_CHECK, done1)            \
done1:                                          \
  for (i2 = Q - q; i2 < L - Q; i2 += Q)          \
    {                                            \
    i = 0;                                     \
    num_iters = Q - width;                         \
    while (num_iters > 0) {                       \
      rsc = rbv_base[dsq[i2 + i]] + (i << 5) + myoffset; \
      step()                                      \
      i++;                                        \
      num_iters--;                                \
      }                                             \
      i += i2;                                   \
    convert(step, NO_CHECK, )                                      \
    }  \
  if ((L - i2) < (Q-width)) num_iters = L-i2;        \
  else                  num_iters = Q-width;         \
  i = 0;                                         \
  while (num_iters > 0) {                        \
    rsc = rbv_base[dsq[i2 + i]] + (i << 5) + myoffset;  \
    step()                                       \
    i++;                                         \
    num_iters--;                                 \
  }                                              \
  i+=i2;                                         \
  convert(step, LENGTH_CHECK, done2)             \
done2:                                          \
  return xEv;


__device__ int calc_band_1(char *dsq, int L, int **rbv_base, int Q, int q, int beginv, int xEv, int ** check_array)
{
  CALC(RESET_1, STEP_BANDS_1, CONVERT_1, 1, check_array)
}

__device__ int calc_band_2(char *dsq, int L, int **rbv_base, int Q, int q, int beginv, int xEv, int ** check_array)
{
  CALC(RESET_2, STEP_BANDS_2, CONVERT_2, 2, check_array)
}

__device__ int calc_band_3(char *dsq, int L, int **rbv_base, int Q, int q, int beginv, int xEv, int ** check_array)
{
  CALC(RESET_3, STEP_BANDS_3, CONVERT_3, 3, check_array)
}

__global__ void SSV_cuda(char *dsq, P7_FILTERMX *mx, int L, int Q, P7_OPROFILE *om, int8_t *retval, int **check_array){
  int myoffset, mywarp, myblock, q;
  int **rbv_base, partner_xEv;
  int last_q = 0;
  int xEv = -128;
  int bands;

  myoffset = threadIdx.x; // Expect a one-dimensional block of 32 threads (one warp)
  mywarp = threadIdx.y; 
  myblock = blockIdx.y;
  int num_reps, i;
  // copy rbv array 
 if((om->M * sizeof(int)) <= 0){
    if((myoffset < 21) && (mywarp == 0)){ // update this for different alphabets
      memcpy((rbv_block + (2048) * myoffset), om->rbv[myoffset], 128*Q);
      rbv_shared[myoffset] = &(rbv_block[myoffset * 2048]);
    }
    rbv_base = (int **) rbv_shared;
  }
  else{
    rbv_base = (int **) om->rbv;
  } 
  __syncthreads(); // Wait until thread 0 done with copy
 
  //int(*fs[MAX_BAND_WIDTH + 1]) (char *, int, int **, int, int, int, int)
   // = {NULL , calc_band_1};

  for(num_reps = 0; num_reps < NUM_REPS; num_reps++){
  last_q = 0; // reset this at start of filter
  xEv = -128;
  /* Use the highest number of bands but no more than MAX_BANDS */
  bands = (Q + MAX_BAND_WIDTH - 1) / MAX_BAND_WIDTH;
  for (i = 0; i < bands; i++) 
    {
      q      = (Q * (i + 1)) / bands;
      switch(q-last_q){
        case 1:
          xEv = calc_band_1(dsq, L, rbv_base, Q, last_q, -128, xEv, check_array);
          break;
        case 2:
          xEv = calc_band_2(dsq, L, rbv_base, Q, last_q, -128, xEv, check_array);
          break; 
        case 3:
          xEv = calc_band_3(dsq, L, rbv_base, Q, last_q, -128, xEv, check_array);
          break;  
        default:
          printf("Illegal band width %d\n", q-last_q);
      }

      last_q = q;
    }

    // Find max of the hvs
    partner_xEv = __shfl_down(xEv, 16); 
    if(myoffset < 16){ // only bottom half of the cores continue from here
  

      xEv = max(xEv, partner_xEv);

      // Reduce 6 4x8-bit quantities to 8
      partner_xEv = __shfl_down(xEv, 8); 
      if(myoffset < 8){ // only bottom half of the cores continue from here
  

        xEv = max(xEv, partner_xEv);

        // Reduce 8 4x8-bit quantities to 4
        partner_xEv = __shfl_down(xEv, 4); 
        if(myoffset < 4){ // only bottom half of the cores continue from here

          xEv = max(xEv, partner_xEv);

          // Reduce 4 4x8-bit quantities to 2
          partner_xEv = __shfl_down(xEv, 2); 
          if(myoffset < 2){ // only bottom half of the cores continue from here

            xEv = max(xEv, partner_xEv);
            // Reduce 2 4x8-bit quantities to 1

            partner_xEv = __shfl_down(xEv, 1); 
            if(myoffset < 1){ // only bottom half of the cores continue from here

              xEv = max(xEv, partner_xEv);

              if((myblock == 0) &&(mywarp ==0) && (myoffset == 0)){ // only one thread writes result

                if (xEv > 127){
                  *retval = 127; 
                }
                else{
                  if (xEv < -128){  
                    *retval = -128;
                  }
                  else{
                    *retval = xEv & 255;
                  }                 
                }
              } 
            }
          }
        }
      }
    }
  }
  return; 
}  

__global__ void SSV_cuda_32bit_memory(char *dsq, P7_FILTERMX *mx, int L, int Q, P7_OPROFILE *om, int8_t *retval){
  int myoffset, mywarp, myblock, q;
  int *rbv, mpv, hv, sv, **rbv_base, sv_shuffle, *dp, partner_hv;
  int neginfmask = -128;
  myoffset = threadIdx.x; // Expect a one-dimensional block of 32 threads (one warp)
  mywarp = threadIdx.y; 
  myblock = blockIdx.y;
  int num_reps, i;
  // copy rbv array 
 if((om->M * sizeof(int)) <= 2048){
    if((myoffset < 21) && (mywarp == 0)){ // update this for different alphabets
      memcpy((rbv_block + (2048) * myoffset), om->rbv[myoffset], 128*Q);
      rbv_shared[myoffset] = &(rbv_block[myoffset * 2048]);
    }
    rbv_base = (int **) rbv_shared;
  }
  else{
    rbv_base = (int **) om->rbv;
  } 
  __syncthreads();


  dp = (int *) malloc(Q * sizeof(int)); // allocate my local roow buffer

  __syncthreads(); //barrier until thread 0 done with any memory setup
 
  for(num_reps = 0; num_reps < NUM_REPS; num_reps++){
    for(i = 0; i < Q; i++){ // initialize our row buffer
      dp[i] = neginfmask;
    }

    mpv = neginfmask;
    hv = neginfmask;
    for(i = 0; i < L; i++){

      rbv = rbv_base[dsq[i]];
      for(q = 0; q < Q; q++){
          sv    = mpv+ rbv[(q * 32) + myoffset];
#ifdef TRUNCATE
          if (sv < -128){
            sv = -128;
          }
#endif
          hv = max(hv, sv);
          mpv   = dp[q];
          dp[q] = sv;
      }
      // Shift SV up one core for next row
      sv_shuffle = __shfl_up(sv, 1);
      if(myoffset == 0){ 
        mpv = neginfmask;
      }
      else{
      mpv = sv_shuffle;
      }
    }
    // Find max of the hvs
    partner_hv = __shfl_down(hv, 16); 
    if(myoffset < 16){ // only bottom half of the cores continue from here
  

      hv = max(hv, partner_hv);

      // Reduce 6 4x8-bit quantities to 8
      partner_hv = __shfl_down(hv, 8); 
      if(myoffset < 8){ // only bottom half of the cores continue from here
  

        hv = max(hv, partner_hv);

        // Reduce 8 4x8-bit quantities to 4
        partner_hv = __shfl_down(hv, 4); 
        if(myoffset < 4){ // only bottom half of the cores continue from here

          hv = max(hv, partner_hv);

          // Reduce 4 4x8-bit quantities to 2
          partner_hv = __shfl_down(hv, 2); 
          if(myoffset < 2){ // only bottom half of the cores continue from here

            hv = max(hv, partner_hv);
            // Reduce 2 4x8-bit quantities to 1

            partner_hv = __shfl_down(hv, 1); 
            if(myoffset < 1){ // only bottom half of the cores continue from here

              hv = max(hv, partner_hv);

              if((myblock == 0) &&(mywarp ==0) && (myoffset == 0)){ // only one thread writes result
                if (hv > 127){
                  *retval = 127; 
                }
                else{
                  if (hv < -128){  
                    *retval = -128;
                  }
                  else{
                    *retval = hv & 255;
                  }                 
                }
              } 
            }
          }
        }
      }
    }
  }
  free(dp);
  return; 
}  

// This attempt uses 32-bit math and DP registers to see if that speeds things up any
__global__ void SSV_cuda_8to32(char *dsq, P7_FILTERMX *mx, int L, int Q, P7_OPROFILE *om, int8_t *retval){
  int myoffset, mywarp, myblock; // per-thread offset from start of each "vector"
  int *rbv, mpv[4], hv[4], hv_max; 
  int neginfmask = -128;
  //unsigned int neginfmask = 0x80808080;
  myoffset = threadIdx.x; // Expect a one-dimensional block of 32 threads (one warp)
  mywarp = threadIdx.y; 
  myblock = blockIdx.y;
  int num_reps, i;
  int dp_local[128];
  int sv[4], sv_shuffle;
  int **rbv_base;
  // copy rbv array 
  if(om->M <= 1024){
    if((myoffset < 21) && (mywarp == 0)){ // update this for different alphabets
      memcpy((rbv_block + 1024 * myoffset), om->rbv[myoffset], 128*Q);
      rbv_shared[myoffset] = &(rbv_block[myoffset * 1024]);
    }
    rbv_base = (int **) rbv_shared;
  }
  else{
    rbv_base = (int **) om->rbv;
  }
  __syncthreads();

  if(myoffset == 0){
    // only do memory setup on one thread
    if(mx->allocM < om->M){
      if(mx->allocM == 0){
        mx->dp = (int16_t *) malloc(512 * Q);
      }
      else{
        free(mx->dp);
          mx->dp = (int16_t *) malloc(512 * Q);
      }
      mx->allocM = om->M;
    }
  }
  __syncthreads(); //barrier until thread 0 done with any memory setup
    

  for(num_reps = 0; num_reps < NUM_REPS; num_reps++){

/*    for(i = 0; i < Q; i++){ // initialize our row buffer
      ((int *)mx->dp)[(i * 128) + (myoffset * 4)] = neginfmask;
      ((int *)mx->dp)[(i * 128) + (myoffset * 4) + 1] = neginfmask;
      ((int *)mx->dp)[(i * 128) + (myoffset * 4) + 2] = neginfmask;
      ((int *)mx->dp)[(i * 128) + (myoffset * 4) + 3] = neginfmask;
    } */
    dp_local[0] = neginfmask;
    dp_local[1] = neginfmask;
    dp_local[2] = neginfmask;
    dp_local[3] = neginfmask;
    dp_local[4] = neginfmask;
    dp_local[5] = neginfmask;
    dp_local[6] = neginfmask;
    dp_local[7] = neginfmask;
    dp_local[8] = neginfmask;
    dp_local[9] = neginfmask;
    dp_local[10] = neginfmask;
    dp_local[11] = neginfmask;

    mpv[0] = neginfmask;
    hv[0] = neginfmask;
    mpv[1] = neginfmask;
    hv[1] = neginfmask;
    mpv[2] = neginfmask;
    hv[2] = neginfmask;
    mpv[3] = neginfmask;
    hv[3] = neginfmask;
    char4 dsq_chunk;
    int *rbv1, *rbv2, *rbv3;
    if(Q == 3){
      char4 rbv_test;
      if(myoffset == 0){
        dsq_buffer[mywarp] = *((int *) dsq); // Grab chunks of dsq four bytes at a time to reduce number of global memory loads
      }
      for(i = 0; (L-i) > 3; i+=4){
        dsq_chunk = *((char4 *)(dsq_buffer+mywarp)) ;
        if(myoffset == 0){
          dsq_buffer[mywarp] = *((int *) (dsq+ i)); // Grab chunks of dsq four bytes at a time to reduce number of global memory loads
        }
        rbv = (rbv_base[dsq_chunk.x]) + myoffset;
        rbv1 =  (rbv_base[dsq_chunk.y]) + myoffset;
        rbv2 = (rbv_base[dsq_chunk.z]) + myoffset;
        rbv3 = (rbv_base[dsq_chunk.w]) + myoffset;
        // row 1
        rbv_test = (* (char4*)rbv); 
        sv[0] = mpv[0] + rbv_test.x; 
#ifdef TRUNCATE
        if(sv[0] < -128){
          sv[0] = -128;
        }
#endif
        MAX(hv[0], hv[0], sv[0]);
        //hv[0] = max(hv[0], sv[0]); 
        mpv[0] = dp_local[0]; 
        dp_local[0] = sv[0];  
        sv[1] = mpv[1] + rbv_test.y;   
#ifdef TRUNCATE
        if(sv[1] < -128){  
          sv[1] = -128; 
        } 
#endif  
        MAX(hv[1], hv[1], sv[1]);
//        hv[1] = max(hv[1], sv[1]); 
        mpv[1] = dp_local[1]; 
        dp_local[1] = sv[1]; 
        sv[2] = mpv[2] + rbv_test.z; 
#ifdef TRUNCATE
        if (sv[2] < -128){ 
          sv[2] = -128;
        }
#endif
        MAX(hv[2], sv[2], hv[2]);        
//        hv[2] = max(hv[2], sv[2]);
        mpv[2] = dp_local[2];
        dp_local[2] = sv[2];
        sv[3] = mpv[3] + rbv_test.w;
#ifdef TRUNCATE
        if (sv[3] < -128){
          sv[3] = -128;
        } 
#endif
//        hv[3] = max(hv[3], sv[3]);
        MAX(hv[3], sv[3], hv[3]);
        mpv[3] = dp_local[3];
        dp_local[3] = sv[3];
        rbv += 32;
        rbv_test = (* (char4*)rbv);
        sv[0] = mpv[0] + rbv_test.x;
#ifdef TRUNCATE        
        if(sv[0] < -128){
          sv[0] = -128;
        } 
#endif  
        MAX(hv[0], sv[0], hv[0]);      
//        hv[0] = max(hv[0], sv[0]);
        mpv[0] = dp_local[4];
        dp_local[4] = sv[0];
        sv[1] = mpv[1] + rbv_test.y;
#ifdef TRUNCATE
        if(sv[1] < -128){
          sv[1] = -128;
        } 
#endif
        MAX(hv[1], sv[1], hv[1]);
//        hv[1] = max(hv[1], sv[1]);
        mpv[1] = dp_local[5];
        dp_local[5] = sv[1];
        sv[2] = mpv[2] + rbv_test.z;
#ifdef TRUNCATE        
        if (sv[2] < -128){
          sv[2] = -128;
        } 
#endif
        MAX(hv[2], hv[2], sv[2]);
//        hv[2] = max(hv[2], sv[2]);
        mpv[2] = dp_local[6];
        dp_local[6] = sv[2];
        sv[3] = mpv[3] + rbv_test.w;
#ifdef TRUNCATE
        if (sv[3] < -128){
          sv[3] = -128;
        }
#endif
        MAX(hv[3], sv[3], hv[3]);
//        hv[3] = max(hv[3], sv[3]);
        mpv[3] = dp_local[7];
        dp_local[7] = sv[3];
        rbv += 32;
        rbv_test = (* (char4*)rbv);
        sv[0] = mpv[0] + rbv_test.x;
#ifdef TRUNCATE
        if(sv[0] < -128){
          sv[0] = -128;
        } 
#endif
        MAX(hv[0], hv[0], sv[0]);
//        hv[0] = max(hv[0], sv[0]);
        mpv[0] = dp_local[8];
        dp_local[8] = sv[0];
        sv[1] = mpv[1] + rbv_test.y;
#ifdef TRUNCATE
        if(sv[1] < -128){
          sv[1] = -128;
        }
#endif
        MAX(hv[1], sv[1], hv[1]);
//        hv[1] = max(hv[1], sv[1]);
        mpv[1] = dp_local[9];
        dp_local[9] = sv[1];
        sv[2] = mpv[2] + rbv_test.z;
#ifdef TRUNCATE
        if (sv[2] < -128){
          sv[2] = -128;
        }
#endif
        MAX(hv[2], sv[2], hv[2]);
//        hv[2] = max(hv[2], sv[2]);
        mpv[2] = dp_local[10];
        dp_local[10] = sv[2];
        sv[3] = mpv[3] + rbv_test.w;
#ifdef TRUNCATE
        if (sv[3] < -128){
          sv[3] = -128;
        }
#endif
        MAX(hv[3], sv[3], hv[3]);
//        hv[3] = max(hv[3], sv[3]);
        mpv[3] = dp_local[11];
        dp_local[11] = sv[3];
        sv_shuffle = __shfl_up(sv[3], 1);
        mpv[3] = sv[2];
        mpv[2] = sv[1];
        mpv[1] = sv[0];
        if(myoffset == 0){ 
          mpv[0] = neginfmask;
        }
        else{
          mpv[0] = sv_shuffle;
        }

        //row 2
        //rbv = (rbv_base[dsq_chunk.y]) + myoffset;
        rbv_test = (* (char4*)rbv1);
        sv[0] = mpv[0] + rbv_test.x;
#ifdef TRUNCATE
        if(sv[0] < -128){
          sv[0] = -128;
        }
#endif
        MAX(hv[0], sv[0], hv[0]);
//        hv[0] = max(hv[0], sv[0]);
        mpv[0] = dp_local[0];
        dp_local[0] = sv[0];
        sv[1] = mpv[1] + rbv_test.y;
#ifdef TRUNCATE
        if(sv[1] < -128){
          sv[1] = -128;
        }
#endif
        MAX(hv[1], sv[1], hv[1]);
//        hv[1] = max(hv[1], sv[1]);
        mpv[1] = dp_local[1];
        dp_local[1] = sv[1];
        sv[2] = mpv[2] + rbv_test.z;
#ifdef TRUNCATE
        if (sv[2] < -128){
          sv[2] = -128;
        }
#endif
        MAX(hv[2], sv[2], hv[2]);
//        hv[2] = max(hv[2], sv[2]);
        mpv[2] = dp_local[2];
        dp_local[2] = sv[2];
        sv[3] = mpv[3] + rbv_test.w;
#ifdef TRUNCATE
        if (sv[3] < -128){
          sv[3] = -128;
        }
#endif
        MAX(hv[3], sv[3], hv[3]);
//        hv[3] = max(hv[3], sv[3]);
        mpv[3] = dp_local[3];
        dp_local[3] = sv[3];
        rbv1 += 32;
        rbv_test = (* (char4*)rbv1);
        sv[0] = mpv[0] + rbv_test.x;
#ifdef TRUNCATE
        if(sv[0] < -128){
          sv[0] = -128;
        }
#endif
        MAX(hv[0], sv[0], hv[0]);
//        hv[0] = max(hv[0], sv[0]);
        mpv[0] = dp_local[4];
        dp_local[4] = sv[0];
        sv[1] = mpv[1] + rbv_test.y;
#ifdef TRUNCATE        
        if(sv[1] < -128){
          sv[1] = -128; 
        }
#endif
        MAX(hv[1], sv[1], hv[1]);
//        hv[1] = max(hv[1], sv[1]);
        mpv[1] = dp_local[5];
        dp_local[5] = sv[1];
        sv[2] = mpv[2] + rbv_test.z;
#ifdef TRUNCATE
        if (sv[2] < -128){
          sv[2] = -128;
        }
#endif
        MAX(hv[2], sv[2], hv[2]);
        //hv[2] = max(hv[2], sv[2]);
        mpv[2] = dp_local[6];
        dp_local[6] = sv[2];
        sv[3] = mpv[3] + rbv_test.w;
#ifdef TRUNCATE
        if (sv[3] < -128){
          sv[3] = -128;
        }
#endif
        MAX(hv[3], sv[3], hv[3]);
//        hv[3] = max(hv[3], sv[3]);
        mpv[3] = dp_local[7];
        dp_local[7] = sv[3];
        rbv1 += 32;
        rbv_test = (* (char4*)rbv1);
        sv[0] = mpv[0] + rbv_test.x;
#ifdef TRUNCATE
        if(sv[0] < -128){
          sv[0] = -128;
        }
#endif
        MAX(hv[0], sv[0], hv[0]);
        //hv[0] = max(hv[0], sv[0]);
        mpv[0] = dp_local[8];
        dp_local[8] = sv[0];
        sv[1] = mpv[1] + rbv_test.y;
#ifdef TRUNCATE
        if(sv[1] < -128){
          sv[1] = -128;
        }
#endif
        MAX(hv[1], sv[1], hv[1]);
        //hv[1] = max(hv[1], sv[1]);
        mpv[1] = dp_local[9];
        dp_local[9] = sv[1];
        sv[2] = mpv[2] + rbv_test.z;
#ifdef TRUNCATE
        if (sv[2] < -128){
          sv[2] = -128;
        }
#endif
        MAX(hv[2], sv[2], hv[2]);
//        hv[2] = max(hv[2], sv[2]);
        mpv[2] = dp_local[10];
        dp_local[10] = sv[2];
        sv[3] = mpv[3] + rbv_test.w;
#ifdef TRUNCATE
        if (sv[3] < -128){
          sv[3] = -128;
        }
#endif
          MAX(hv[3], sv[3], hv[3]);
//        hv[3] = max(hv[3], sv[3]);
        mpv[3] = dp_local[11];
        dp_local[11] = sv[3];
        sv_shuffle = __shfl_up(sv[3], 1);
        mpv[3] = sv[2];
        mpv[2] = sv[1];
        mpv[1] = sv[0];
        if(myoffset == 0){ 
          mpv[0] = neginfmask;
        }
        else{
          mpv[0] = sv_shuffle;
        }

        // Row 3
        //rbv = (rbv_base[dsq_chunk.z]) + myoffset; 
        
        rbv_test = (* (char4*)rbv2);
        sv[0] = mpv[0] + rbv_test.x;
#ifdef TRUNCATE
        if(sv[0] < -128){
          sv[0] = -128;
        }
#endif
        MAX(hv[0], sv[0], hv[0]);
        //hv[0] = max(hv[0], sv[0]);
        mpv[0] = dp_local[0];
        dp_local[0] = sv[0];
        sv[1] = mpv[1] + rbv_test.y;
#ifdef TRUNCATE
        if(sv[1] < -128){
          sv[1] = -128;
        }
#endif
        MAX(hv[1], sv[1], hv[1]);
        //hv[1] = max(hv[1], sv[1]);
        mpv[1] = dp_local[1];
        dp_local[1] = sv[1];
        sv[2] = mpv[2] + rbv_test.z;
#ifdef TRUNCATE
        if (sv[2] < -128){
          sv[2] = -128;
        }
#endif
        MAX(hv[2], sv[2], hv[2]);
        //hv[2] = max(hv[2], sv[2]);
        mpv[2] = dp_local[2];
        dp_local[2] = sv[2];
        sv[3] = mpv[3] + rbv_test.w;
#ifdef TRUNCATE
        if (sv[3] < -128){
          sv[3] = -128;
        }
#endif
        MAX(hv[3], sv[3], hv[3]);
//        hv[3] = max(hv[3], sv[3]);
        mpv[3] = dp_local[3];
        dp_local[3] = sv[3];
        rbv2 += 32;
        rbv_test = (* (char4*)rbv2);
        sv[0] = mpv[0] + rbv_test.x;
#ifdef TRUNCATE
        if(sv[0] < -128){
          sv[0] = -128;
        }
#endif        
        MAX(hv[0], sv[0], hv[0]);
        //hv[0] = max(hv[0], sv[0]);
        mpv[0] = dp_local[4];
        dp_local[4] = sv[0];
        sv[1] = mpv[1] + rbv_test.y;
#ifdef  TRUNCATE    
        if(sv[1] < -128){
          sv[1] = -128;
        }
#endif
        MAX(hv[1], sv[1], hv[1]);
//        hv[1] = max(hv[1], sv[1]);
        mpv[1] = dp_local[5];
        dp_local[5] = sv[1];
        sv[2] = mpv[2] + rbv_test.z;
#ifdef TRUNCATE
        if (sv[2] < -128){
          sv[2] = -128;
        }
#endif        
        MAX(hv[2], sv[2], hv[2]);
//        hv[2] = max(hv[2], sv[2]);
        mpv[2] = dp_local[6];
        dp_local[6] = sv[2];
        sv[3] = mpv[3] + rbv_test.w;
#ifdef  TRUNCATE
        if (sv[3] < -128){
          sv[3] = -128; 
        }
#endif
        MAX(hv[3], sv[3], hv[3]);
        //hv[3] = max(hv[3], sv[3]);
        mpv[3] = dp_local[7];
        dp_local[7] = sv[3];
        rbv2 += 32; 
        rbv_test = (* (char4*)rbv2);
        sv[0] = mpv[0] + rbv_test.x;
#ifdef TRUNCATE
        if(sv[0] < -128){
          sv[0] = -128; 
        } 
#endif
        MAX(hv[0], sv[0], hv[0]);
        hv[0] = max(hv[0], sv[0]);
        mpv[0] = dp_local[8];
        dp_local[8] = sv[0];
        sv[1] = mpv[1] + rbv_test.y;
#ifdef TRUNCATE
        if(sv[1] < -128){
          sv[1] = -128; 
        }
#endif
        MAX(hv[1], sv[1], hv[1]);
//        hv[1] = max(hv[1], sv[1]);
        mpv[1] = dp_local[9];
        dp_local[9] = sv[1];
        sv[2] = mpv[2] + rbv_test.z;
#ifdef TRUNCATE
        if (sv[2] < -128){
          sv[2] = -128;
        }
#endif
        MAX(hv[2], sv[2], hv[2]);
//        hv[2] = max(hv[2], sv[2]);
        mpv[2] = dp_local[10];
        dp_local[10] = sv[2];
        sv[3] = mpv[3] + rbv_test.w;
#ifdef TRUNCATE
        if (sv[3] < -128){
          sv[3] = -128;
        }
#endif
        MAX(hv[3], sv[3], hv[3]);
//        hv[3] = max(hv[3], sv[3]);
        mpv[3] = dp_local[11];
        dp_local[11] = sv[3];
        sv_shuffle = __shfl_up(sv[3], 1);
        mpv[3] = sv[2];
        mpv[2] = sv[1];
        mpv[1] = sv[0];
        if(myoffset == 0){ 
          mpv[0] = neginfmask;
        }
        else{
          mpv[0] = sv_shuffle; 
        }
        //row 4
        //rbv = (rbv_base[dsq_chunk.w]) + myoffset; 
        
        rbv_test = (* (char4*)rbv3);
        sv[0] = mpv[0] + rbv_test.x;
#ifdef TRUNCATE
        if(sv[0] < -128){
          sv[0] = -128;
        }
#endif
        MAX(hv[0], sv[0], hv[0]);
        //hv[0] = max(hv[0], sv[0]); 
        mpv[0] = dp_local[0]; 
        dp_local[0] = sv[0];  
        sv[1] = mpv[1] + rbv_test.y;
#ifdef TRUNCATE
        if(sv[1] < -128){
          sv[1] = -128; 
        }
#endif
        MAX(hv[1], sv[1], hv[1]);
//        hv[1] = max(hv[1], sv[1]);
        mpv[1] = dp_local[1];
        dp_local[1] = sv[1];
        sv[2] = mpv[2] + rbv_test.z;
#ifdef  TRUNCATE
        if (sv[2] < -128){
          sv[2] = -128;
        }
#endif
        MAX(hv[2], sv[2], hv[2]);
//        hv[2] = max(hv[2], sv[2]);
        mpv[2] = dp_local[2];
        dp_local[2] = sv[2];
        sv[3] = mpv[3] + rbv_test.w;
#ifdef  TRUNCATE
        if (sv[3] < -128){
          sv[3] = -128; 
        } 
#endif
        MAX(hv[3], sv[3], hv[3]);
        //hv[3] = max(hv[3], sv[3]);
        mpv[3] = dp_local[3]; 
        dp_local[3] = sv[3];
        rbv3 += 32;
        rbv_test = (* (char4*)rbv3);
        sv[0] = mpv[0] + rbv_test.x;
#ifdef TRUNCATE
        if(sv[0] < -128){
          sv[0] = -128; 
        }
#endif
           MAX(hv[0], sv[0], hv[0]);
        //hv[0] = max(hv[0], sv[0]);
        mpv[0] = dp_local[4];
        dp_local[4] = sv[0];
        sv[1] = mpv[1] + rbv_test.y;
#ifdef TRUNCATE
        if(sv[1] < -128){
          sv[1] = -128;
        }
#endif        
        MAX(hv[1], sv[1], hv[1]);
        //hv[1] = max(hv[1], sv[1]);
        mpv[1] = dp_local[5];
        dp_local[5] = sv[1];
        sv[2] = mpv[2] + rbv_test.z;
#ifdef TRUNCATE
        if (sv[2] < -128){
          sv[2] = -128;
        }
#endif  
        MAX(hv[2], sv[2], hv[2]);      
        //hv[2] = max(hv[2], sv[2]);
        mpv[2] = dp_local[6];
        dp_local[6] = sv[2];
        sv[3] = mpv[3] + rbv_test.w;
#ifdef TRUNCATE
        if (sv[3] < -128){
          sv[3] = -128;
        }
#endif
        MAX(hv[3], sv[3], hv[3]);
        //hv[3] = max(hv[3], sv[3]);
        mpv[3] = dp_local[7];
        dp_local[7] = sv[3];
        rbv3 += 32;
        rbv_test = (* (char4*)rbv3); 
        sv[0] = mpv[0] + rbv_test.x;
#ifdef TRUNCATE
        if(sv[0] < -128){
          sv[0] = -128;
        }
#endif
        MAX(hv[0], sv[0], hv[0]);
        //hv[0] = max(hv[0], sv[0]);
        mpv[0] = dp_local[8];
        dp_local[8] = sv[0];
        sv[1] = mpv[1] + rbv_test.y;
#ifdef TRUNCATE
        if(sv[1] < -128){
          sv[1] = -128; 
        }
#endif
        MAX(hv[1], sv[1], hv[1]);
//        hv[1] = max(hv[1], sv[1]);
        mpv[1] = dp_local[9];
        dp_local[9] = sv[1];
        sv[2] = mpv[2] + rbv_test.z;
#ifdef TRUNCATE
        if (sv[2] < -128){
          sv[2] = -128;
        }
#endif
        MAX(hv[2], sv[2], hv[2]);
//        hv[2] = max(hv[2], sv[2]);
        mpv[2] = dp_local[10];
        dp_local[10] = sv[2];
        sv[3] = mpv[3] + rbv_test.w;
#ifdef TRUNCATE
        if (sv[3] < -128){
          sv[3] = -128; 
        }
#endif
        MAX(hv[3], sv[3], hv[3]);
//        hv[3] = max(hv[3], sv[3]);
        mpv[3] = dp_local[11];
        dp_local[11] = sv[3];
        sv_shuffle = __shfl_up(sv[3], 1);
        mpv[3] = sv[2];
        mpv[2] = sv[1];
        mpv[1] = sv[0];
        if(myoffset == 0){
          mpv[0] = neginfmask;
        }
        else{
          mpv[0] = sv_shuffle;
        }
      }
      dsq_chunk = *((char4 *) dsq_buffer+mywarp);
      int remaining_rows = L-i;
      if(remaining_rows > 0){
        // If we get in here, there must be at least one row remaining to compute
        rbv = (rbv_base[dsq_chunk.x]) + myoffset;  
        // row 1
        rbv_test = (* (char4*)rbv);
        sv[0] = mpv[0] + rbv_test.x;
#ifdef TRUNCATE
        if(sv[0] < -128){
          sv[0] = -128;
        }
#endif
        MAX(hv[0], sv[0], hv[0]);
        //hv[0] = max(hv[0], sv[0]);
        mpv[0] = dp_local[0];
        dp_local[0] = sv[0]; 
        sv[1] = mpv[1] + rbv_test.y;
#ifdef TRUNCATE
        if(sv[1] < -128){ 
          sv[1] = -128; 
        }
#endif
        MAX(hv[1], sv[1], hv[1]);
        //hv[1] = max(hv[1], sv[1]); 
        mpv[1] = dp_local[1];
        dp_local[1] = sv[1]; 
        sv[2] = mpv[2] + rbv_test.z;
#ifdef TRUNCATE
        if (sv[2] < -128){
          sv[2] = -128; 
        } 
#endif
        MAX(hv[2], sv[2], hv[2]);
//        hv[2] = max(hv[2], sv[2]);
        mpv[2] = dp_local[2]; 
        dp_local[2] = sv[2]; 
        sv[3] = mpv[3] + rbv_test.w;
#ifdef TRUNCATE
        if (sv[3] < -128){
          sv[3] = -128;
        } 
#endif
        MAX(hv[3], sv[3], hv[3]);    
//        hv[3] = max(hv[3], sv[3]);
        mpv[3] = dp_local[3];
        dp_local[3] = sv[3];
        rbv += 32;
        rbv_test = (* (char4*)rbv);
        sv[0] = mpv[0] + rbv_test.x;
#ifdef TRUNCATE 
        if(sv[0] < -128){ 
          sv[0] = -128;
        }
#endif        
        MAX(hv[0], sv[0], hv[0]);
 //       hv[0] = max(hv[0], sv[0]);
        mpv[0] = dp_local[4];
        dp_local[4] = sv[0];
        sv[1] = mpv[1] + rbv_test.y;
#ifdef TRUNCATE
        if(sv[1] < -128){
          sv[1] = -128;
        }
#endif
        MAX(hv[1], sv[1], hv[1]);
//        hv[1] = max(hv[1], sv[1]);
        mpv[1] = dp_local[5];
        dp_local[5] = sv[1];
        sv[2] = mpv[2] + rbv_test.z;
#ifdef TRUNCATE
        if (sv[2] < -128){ 
          sv[2] = -128;
        }
#endif
        MAX(hv[2], sv[2], hv[2]);
//        hv[2] = max(hv[2], sv[2]);
        mpv[2] = dp_local[6];
        dp_local[6] = sv[2];
        sv[3] = mpv[3] + rbv_test.w;
#ifdef TRUNCATE
        if (sv[3] < -128){
          sv[3] = -128;
        }
#endif
        MAX(hv[3], sv[3], hv[3]);
//        hv[3] = max(hv[3], sv[3]);
        mpv[3] = dp_local[7];
        dp_local[7] = sv[3];
        rbv += 32;
        rbv_test = (* (char4*)rbv);
        sv[0] = mpv[0] + rbv_test.x;
#ifdef TRUNCATE
        if(sv[0] < -128){
          sv[0] = -128; 
        }
#endif
        MAX(hv[0], sv[0], hv[0]);
//        hv[0] = max(hv[0], sv[0]);
        mpv[0] = dp_local[8];
        dp_local[8] = sv[0];
        sv[1] = mpv[1] + rbv_test.y;
#ifdef TRUNCATE
        if(sv[1] < -128){
          sv[1] = -128;
        }
#endif
        MAX(hv[1], sv[1], hv[1]);
//        hv[1] = max(hv[1], sv[1]);
        mpv[1] = dp_local[9];
        dp_local[9] = sv[1];
        sv[2] = mpv[2] + rbv_test.z;
#ifdef TRUNCATE
        if (sv[2] < -128){
          sv[2] = -128;
        }
#endif
        MAX(hv[2], sv[2], hv[2]);
//        hv[2] = max(hv[2], sv[2]);
        mpv[2] = dp_local[10];
        dp_local[10] = sv[2];
        sv[3] = mpv[3] + rbv_test.w;
#ifdef TRUNCATE        
        if (sv[3] < -128){
          sv[3] = -128;
        }
#endif
        MAX(hv[3], sv[3], hv[3]);
//        hv[3] = max(hv[3], sv[3]);
        mpv[3] = dp_local[11];
        dp_local[11] = sv[3];
        rbv += 32; 
        sv_shuffle = __shfl_up(sv[3], 1);
        mpv[3] = sv[2];
        mpv[2] = sv[1];
        mpv[1] = sv[0];
        if(myoffset == 0){
          mpv[0] = neginfmask;
        }
        else{
          mpv[0] = sv_shuffle;
        }
        if(remaining_rows > 1){
          //row 2
          rbv = (rbv_base[dsq_chunk.y]) + myoffset;
          rbv_test = (* (char4*)rbv);
          sv[0] = mpv[0] + rbv_test.x;
#ifdef TRUNCATE
          if(sv[0] < -128){
            sv[0] = -128;
          }
#endif
          MAX(hv[0], sv[0], hv[0]);
          //hv[0] = max(hv[0], sv[0]);
          mpv[0] = dp_local[0];
          dp_local[0] = sv[0]; 
          sv[1] = mpv[1] + rbv_test.y;
#ifdef TRUNCATE
          if(sv[1] < -128){
            sv[1] = -128;
          }
#endif    
          MAX(hv[1], sv[1], hv[1]);     
        //  hv[1] = max(hv[1], sv[1]);
          mpv[1] = dp_local[1];
          dp_local[1] = sv[1];
          sv[2] = mpv[2] + rbv_test.z; 
#ifdef TRUNCATE
          if (sv[2] < -128){ 
            sv[2] = -128; 
          }
#endif
          MAX(hv[2], sv[2], hv[2]);
          //hv[2] = max(hv[2], sv[2]);
          mpv[2] = dp_local[2];
          dp_local[2] = sv[2];
          sv[3] = mpv[3] + rbv_test.w;
#ifdef TRUNCATE
          if (sv[3] < -128){ 
            sv[3] = -128;
          }
#endif
          MAX(hv[3], sv[3], hv[3]);
          hv[3] = max(hv[3], sv[3]);
          mpv[3] = dp_local[3];
          dp_local[3] = sv[3];
          rbv += 32;
          rbv_test = (* (char4*)rbv);
          sv[0] = mpv[0] + rbv_test.x;
#ifdef TRUNCATE
          if(sv[0] < -128){
            sv[0] = -128;
          }
#endif
          MAX(hv[0], sv[0], hv[0]);
//          hv[0] = max(hv[0], sv[0]);
          mpv[0] = dp_local[4];
          dp_local[4] = sv[0];
          sv[1] = mpv[1] + rbv_test.y;
#ifdef TRUNCATE
          if(sv[1] < -128){
            sv[1] = -128;
          }
#endif
          MAX(hv[1], sv[1], hv[1]);
//          hv[1] = max(hv[1], sv[1]);
          mpv[1] = dp_local[5];
          dp_local[5] = sv[1];
          sv[2] = mpv[2] + rbv_test.z;
#ifdef TRUNCATE
          if (sv[2] < -128){
            sv[2] = -128;
          }
#endif          
          MAX(hv[2], sv[2], hv[2]); 
//          hv[2] = max(hv[2], sv[2]);
          mpv[2] = dp_local[6];
          dp_local[6] = sv[2];
          sv[3] = mpv[3] + rbv_test.w;
#ifdef TRUNCATE
          if (sv[3] < -128){
            sv[3] = -128;
          }
#endif
        MAX(hv[3], sv[3], hv[3]);
//          hv[3] = max(hv[3], sv[3]);
          mpv[3] = dp_local[7];
          dp_local[7] = sv[3];
          rbv += 32;

          rbv_test = (* (char4*)rbv);
          sv[0] = mpv[0] + rbv_test.x;
#ifdef TRUNCATE
          if(sv[0] < -128){ 
            sv[0] = -128; 
          } 
#endif
          MAX(hv[0], sv[0], hv[0]);
          hv[0] = max(hv[0], sv[0]);
          mpv[0] = dp_local[8]; 
          dp_local[8] = sv[0]; 
          sv[1] = mpv[1] + rbv_test.y;
#ifdef TRUNCATE
          if(sv[1] < -128){ 
            sv[1] = -128; 
          }
#endif
          MAX(hv[1], sv[1], hv[1]);
          //hv[1] = max(hv[1], sv[1]); 
          mpv[1] = dp_local[9]; 
          dp_local[9] = sv[1]; 
          sv[2] = mpv[2] + rbv_test.z; 
#ifdef TRUNCATE
          if (sv[2] < -128){ 
            sv[2] = -128; 
          }
#endif
          MAX(hv[2], sv[2], hv[2]);
          //hv[2] = max(hv[2], sv[2]); 
          mpv[2] = dp_local[10]; 
          dp_local[10] = sv[2]; 
          sv[3] = mpv[3] + rbv_test.w; 
#ifdef TRUNCATE
          if (sv[3] < -128){ 
            sv[3] = -128; 
          } 
#endif
          MAX(hv[3], sv[3], hv[3]);
//          hv[3] = max(hv[3], sv[3]); 
          mpv[3] = dp_local[11]; 
          dp_local[11] = sv[3]; 
          rbv += 32;  
          sv_shuffle = __shfl_up(sv[3], 1);  
          mpv[3] = sv[2];
          mpv[2] = sv[1];
          mpv[1] = sv[0];
          if(myoffset == 0){ 
            mpv[0] = neginfmask;
          }
          else{
            mpv[0] = sv_shuffle; 
          }
          if(remaining_rows > 2){
            // Row 3
            rbv = (rbv_base[dsq_chunk.z]) + myoffset; 
        
            rbv_test = (* (char4*)rbv);
            sv[0] = mpv[0] + rbv_test.x; 
#ifdef TRUNCATE
            if(sv[0] < -128){
              sv[0] = -128;
            }
#endif
            MAX(hv[0], sv[0], hv[0]);
  //          hv[0] = max(hv[0], sv[0]);
            mpv[0] = dp_local[0]; 
            dp_local[0] = sv[0];
            sv[1] = mpv[1] + rbv_test.y;
#ifdef TRUNCATE
            if(sv[1] < -128){ 
              sv[1] = -128; 
            } 
#endif
            MAX(hv[1], sv[1], hv[1]);
            //hv[1] = max(hv[1], sv[1]);
            mpv[1] = dp_local[1];
            dp_local[1] = sv[1];
            sv[2] = mpv[2] + rbv_test.z;
#ifdef TRUNCATE
            if (sv[2] < -128){ 
              sv[2] = -128; 
            } 
#endif
            MAX(hv[2], sv[2], hv[2]);
//            hv[2] = max(hv[2], sv[2]); 
            mpv[2] = dp_local[2]; 
            dp_local[2] = sv[2]; 
            sv[3] = mpv[3] + rbv_test.w; 
#ifdef TRUNCATE
            if (sv[3] < -128){ 
              sv[3] = -128; 
            } 
#endif
            MAX(hv[3], sv[3], hv[3]);
  //          hv[3] = max(hv[3], sv[3]);
            mpv[3] = dp_local[3];
            dp_local[3] = sv[3];
            rbv += 32; 
          
            rbv_test = (* (char4*)rbv); 
            sv[0] = mpv[0] + rbv_test.x; 
#ifdef TRUNCATE
            if(sv[0] < -128){ 
              sv[0] = -128; 
            } 
#endif
            MAX(hv[0], sv[0], hv[0]);
            //hv[0] = max(hv[0], sv[0]);
            mpv[0] = dp_local[4];
            dp_local[4] = sv[0]; 
            sv[1] = mpv[1] + rbv_test.y;
#ifdef TRUNCATE            
            if(sv[1] < -128){ 
              sv[1] = -128;
            } 
#endif
            MAX(hv[1], sv[1], hv[1]);
            //hv[1] = max(hv[1], sv[1]);
            mpv[1] = dp_local[5];
            dp_local[5] = sv[1];
            sv[2] = mpv[2] + rbv_test.z;
#ifdef TRUNCATE
            if (sv[2] < -128){ 
              sv[2] = -128; 
            } 
#endif
            MAX(hv[2], sv[2], hv[2]);
//            hv[2] = max(hv[2], sv[2]);
            mpv[2] = dp_local[6];
            dp_local[6] = sv[2]; 
            sv[3] = mpv[3] + rbv_test.w;
#ifdef TRUNCATE
            if (sv[3] < -128){
              sv[3] = -128;
            }
#endif
            MAX(hv[3], sv[3], hv[3]);
         //   hv[3] = max(hv[3], sv[3]);
            mpv[3] = dp_local[7];
            dp_local[7] = sv[3];
            rbv += 32; 
            rbv_test = (* (char4*)rbv);
            sv[0] = mpv[0] + rbv_test.x;
#ifdef TRUNCATE
            if(sv[0] < -128){
              sv[0] = -128; 
            }
#endif
            MAX(hv[0], sv[0], hv[0]);
            //hv[0] = max(hv[0], sv[0]); 
            mpv[0] = dp_local[8]; 
            dp_local[8] = sv[0]; 
            sv[1] = mpv[1] + rbv_test.y;
#ifdef TRUNCATE
            if(sv[1] < -128){ 
              sv[1] = -128; 
            }
#endif
            MAX(hv[1], sv[1], hv[1]);
//            hv[1] = max(hv[1], sv[1]);
            mpv[1] = dp_local[9];
            dp_local[9] = sv[1];
            sv[2] = mpv[2] + rbv_test.z;
#ifdef TRUNCATE
            if (sv[2] < -128){
              sv[2] = -128; 
            }
#endif
            MAX(hv[2], sv[2], hv[2]);
  //          hv[2] = max(hv[2], sv[2]);
            mpv[2] = dp_local[10];
            dp_local[10] = sv[2];
            sv[3] = mpv[3] + rbv_test.w;
#ifdef TRUNCATE            
            if (sv[3] < -128){
              sv[3] = -128;
            } 
#endif            
            MAX(hv[3], sv[3], hv[3]);
            //hv[3] = max(hv[3], sv[3]);
            mpv[3] = dp_local[11]; 
            dp_local[11] = sv[3]; 
            rbv += 32;  
            }
          }
        }
    
    }

    int partner_hv;

    // Done with main loop.  Now reduce answer vector (hv) to one byte for return
    hv_max = max(hv[0], hv[1]);
    hv_max = max(hv_max, hv[2]);
    hv_max = max(hv_max, hv[3]);
    partner_hv = __shfl_down(hv_max, 16); 
    if(myoffset < 16){ // only bottom half of the cores continue from here
  

      hv_max = max(hv_max, partner_hv);

      // Reduce 6 4x8-bit quantities to 8
      partner_hv = __shfl_down(hv_max, 8); 
      if(myoffset < 8){ // only bottom half of the cores continue from here
  

        hv_max = max(hv_max, partner_hv);

        // Reduce 8 4x8-bit quantities to 4
        partner_hv = __shfl_down(hv_max, 4); 
        if(myoffset < 4){ // only bottom half of the cores continue from here

          hv_max = max(hv_max, partner_hv);

          // Reduce 4 4x8-bit quantities to 2
          partner_hv = __shfl_down(hv_max, 2); 
          if(myoffset < 2){ // only bottom half of the cores continue from here

            hv_max = max(hv_max, partner_hv);
            // Reduce 2 4x8-bit quantities to 1

            partner_hv = __shfl_down(hv_max, 1); 
            if(myoffset < 1){ // only bottom half of the cores continue from here

              hv_max = max(hv_max, partner_hv);

              if((myblock == 0) &&(mywarp ==0) && (myoffset == 0)){ // only one thread writes result
                if (hv_max > 127){
                  *retval = 127; 
                }
                else{
                  if (hv_max < -128){  
                    *retval = -128;
                  }
                  else{
                    *retval = hv_max & 255;
                  }                 
                }
              } 
            }
          }
        }
      }
    }
  } 
  return;
}


__global__ void SSV_cuda_packed(char *dsq, P7_FILTERMX *mx, int L, int Q, P7_OPROFILE *om, int8_t *retval){

  int myoffset, mywarp, myblock; // per-thread offset from start of each "vector"
  unsigned int *rbv, *dp, mpv, hv; 
  unsigned int neginfmask = 0x80808080;
  myoffset = threadIdx.x; // Expect a one-dimensional block of 32 threads (one warp)
  mywarp = threadIdx.y; 
  myblock = blockIdx.y;
  int num_reps, i;
  int dp_local[128];
  int sv, sv_shuffle, sv0, sv1, sv2, hv0, hv1, hv2;
  unsigned int **rbv_base;
  // copy rbv array 
  if(om->M <= 1024){
    if((myoffset < 21) && (mywarp == 0)){ // update this for different alphabets
      memcpy((rbv_block + 1024 * myoffset), om->rbv[myoffset], 128*Q);
      rbv_shared[myoffset] = &(rbv_block[myoffset * 1024]);
    }
    rbv_base = (unsigned int **) rbv_shared;
  }
  else{
    rbv_base = (unsigned int **) om->rbv;
  }
  __syncthreads();

  for(num_reps = 0; num_reps < 10000; num_reps++){
    dp = ((unsigned int *)mx->dp) + myoffset;

  mpv = neginfmask;
  hv = neginfmask;
  int dsq_mod;
  int dsq_chunk, dsq_chunk_next;
  unsigned int *rbv0, *rbv1, *rbv2;
  if(Q == 3){
    //all-registers version
    dp_local[0] = neginfmask;
    dp_local[1] = neginfmask;
    dp_local[2] = neginfmask;
    hv0 = neginfmask;
    hv1 = neginfmask;
    hv2 = neginfmask;

    i = 0;
    dsq_chunk_next = *((int *) (dsq+i)); // Grab chunks of dsq four bytes at a time to reduce number of global memory loads
    while((L - i) > 3){ // unroll loop by four
      dsq_chunk = dsq_chunk_next;
      i+=4;
      dsq_chunk_next = *((int *) (dsq+i)); // Try prefetching this
      // 1st iteration
      rbv0 = (rbv_base[dsq_chunk &255]) + myoffset;
      dsq_chunk = dsq_chunk >> 8;
      dsq_mod++;
      // triply-unrolled loop
      sv0 = __vaddss4(mpv, *rbv0);
      hv0 = __vmaxs4(hv0, sv0);
      mpv = dp_local[0];
      dp_local[0] = sv0;
//      rbv += 32;  // advance to next 
      sv1 = __vaddss4(mpv, *(rbv0+32));
      hv1 = __vmaxs4(hv1, sv1);
      mpv = dp_local[1];
      dp_local[1] = sv1;
//      rbv += 32;  // advance to next 
      sv2 = __vaddss4(mpv, *(rbv0+64));
      hv2 = __vmaxs4(hv2, sv2);
      mpv = dp_local[2];
      dp_local[2] = sv2;
//      rbv += 32;  // advance to next 
      sv = sv2;

     // Now, leftshift (memory order) the sv vector to get the next mpv
      sv_shuffle = __shfl_up(sv, 1);  // Get the next core up's value of SV
      if(myoffset == 0){
        mpv = __byte_perm(sv, neginfmask, 0x2107); //left-shifts sv by one byte, puts the high byte of neginfmask in the low byte of sv
      }
      else{
        mpv = __byte_perm(sv, sv_shuffle, 0x2107); //left-shifts sv by one byte, puts the high byte of sv_shuffle in the low byte of sv
      }

      // 2nd iteration
      rbv1 = (rbv_base[dsq_chunk &255]) + myoffset;
      dsq_chunk = dsq_chunk >> 8;
      // triply-unrolled loop
      sv0 = __vaddss4(mpv, *rbv1);
      hv0 = __vmaxs4(hv0, sv0);
      mpv = dp_local[0];
      dp_local[0] = sv0;
//      rbv += 32;  // advance to next 
      sv1 = __vaddss4(mpv, *(rbv1+32));
      hv1 = __vmaxs4(hv1, sv1);
      mpv = dp_local[1];
      dp_local[1] = sv1;
//      rbv += 32;  // advance to next 
      sv2 = __vaddss4(mpv, *(rbv1+64));
      hv2 = __vmaxs4(hv2, sv2);
      mpv = dp_local[2];
      dp_local[2] = sv2;
//      rbv += 32;  // advance to next 
      sv = sv2;

     // Now, leftshift (memory order) the sv vector to get the next mpv
      sv_shuffle = __shfl_up(sv, 1);  // Get the next core up's value of SV
      if(myoffset == 0){
        mpv = __byte_perm(sv, neginfmask, 0x2107); //left-shifts sv by one byte, puts the high byte of neginfmask in the low byte of sv
      }
      else{
        mpv = __byte_perm(sv, sv_shuffle, 0x2107); //left-shifts sv by one byte, puts the high byte of sv_shuffle in the low byte of sv
      }

      //3rd iteration
      rbv2 = (rbv_base[dsq_chunk &255]) + myoffset;
      dsq_chunk = dsq_chunk >> 8;
      // triply-unrolled loop
      sv0 = __vaddss4(mpv, *rbv2);
      hv0 = __vmaxs4(hv0, sv0);
      mpv = dp_local[0];
      dp_local[0] = sv0;
//      rbv += 32;  // advance to next 
      sv1 = __vaddss4(mpv, *(rbv2+32));
      hv1 = __vmaxs4(hv1, sv1);
      mpv = dp_local[1];
      dp_local[1] = sv1;
//      rbv += 32;  // advance to next 
      sv2 = __vaddss4(mpv, *(rbv2+64));
      hv2 = __vmaxs4(hv2, sv2);
      mpv = dp_local[2];
      dp_local[2] = sv2;
//      rbv += 32;  // advance to next 
      sv = sv2;

     // Now, leftshift (memory order) the sv vector to get the next mpv
      sv_shuffle = __shfl_up(sv, 1);  // Get the next core up's value of SV
      if(myoffset == 0){
        mpv = __byte_perm(sv, neginfmask, 0x2107); //left-shifts sv by one byte, puts the high byte of neginfmask in the low byte of sv
      }
      else{
        mpv = __byte_perm(sv, sv_shuffle, 0x2107); //left-shifts sv by one byte, puts the high byte of sv_shuffle in the low byte of sv
      }

      //4th iteration
      rbv = (rbv_base[dsq_chunk &255]) + myoffset;
      dsq_chunk = dsq_chunk >> 8;
      // triply-unrolled loop
      sv0 = __vaddss4(mpv, *rbv);
      hv0 = __vmaxs4(hv0, sv0);
      mpv = dp_local[0];
      dp_local[0] = sv0;
//      rbv += 32;  // advance to next 
      sv1 = __vaddss4(mpv, *(rbv+32));
      hv1 = __vmaxs4(hv1, sv1);
      mpv = dp_local[1];
      dp_local[1] = sv1;
//      rbv += 32;  // advance to next 
      sv2 = __vaddss4(mpv, *(rbv+64));
      hv2 = __vmaxs4(hv2, sv2);
      mpv = dp_local[2];
      dp_local[2] = sv2;
//      rbv += 32;  // advance to next 
      sv = sv2;

     // Now, leftshift (memory order) the sv vector to get the next mpv
      sv_shuffle = __shfl_up(sv, 1);  // Get the next core up's value of SV
      if(myoffset == 0){
        mpv = __byte_perm(sv, neginfmask, 0x2107); //left-shifts sv by one byte, puts the high byte of neginfmask in the low byte of sv
      }
      else{
        mpv = __byte_perm(sv, sv_shuffle, 0x2107); //left-shifts sv by one byte, puts the high byte of sv_shuffle in the low byte of sv
      }

    }

    for(dsq_mod=4; i < L; i++){
      if(dsq_mod == 4){
        dsq_chunk = *((int *) (dsq+i)); // Grab chunks of dsq four bytes at a time to reduce number of global memory loads
        dsq_mod = 0;
      }

      rbv = (rbv_base[dsq_chunk &255]) + myoffset;
      dsq_chunk = dsq_chunk >> 8;
      dsq_mod++;
      int rbv0 = *rbv;
      int rbv1 = *(rbv+32);
      int rbv2 = *(rbv+64);
      // triply-unrolled loop
      sv0 = __vaddss4(mpv, rbv0);
      hv0 = __vmaxs4(hv0, sv0);
      mpv = dp_local[0];
      dp_local[0] = sv0;
//      rbv += 32;  // advance to next 
      sv1 = __vaddss4(mpv, rbv1);
      hv1 = __vmaxs4(hv1, sv1);
      mpv = dp_local[1];
      dp_local[1] = sv1;
//      rbv += 32;  // advance to next 
      sv2 = __vaddss4(mpv, rbv2);
      hv2 = __vmaxs4(hv2, sv2);
      mpv = dp_local[2];
      dp_local[2] = sv2;
//      rbv += 32;  // advance to next 
      sv = sv2;

     // Now, leftshift (memory order) the sv vector to get the next mpv
      sv_shuffle = __shfl_up(sv, 1);  // Get the next core up's value of SV
      if(myoffset == 0){
        mpv = __byte_perm(sv, neginfmask, 0x2107); //left-shifts sv by one byte, puts the high byte of neginfmask in the low byte of sv
      }
      else{
        mpv = __byte_perm(sv, sv_shuffle, 0x2107); //left-shifts sv by one byte, puts the high byte of sv_shuffle in the low byte of sv
      }
    }
    hv = __vmaxs4(hv0, hv1);
    hv = __vmaxs4(hv, hv2);
  }
  else{ // in-memory, slower, version
    
    if(myoffset == 0){
      // only do memory setup on one thread
      if(mx->allocM < om->M){
        if(mx->allocM == 0){
          mx->dp = (int16_t *) malloc(128 * Q);
        }
        else{
          free(mx->dp);
          mx->dp = (int16_t *) malloc(128 * Q);
        }
        mx->allocM = om->M;
      }
    }
    __syncthreads(); //barrier until thread 0 done with any memory setup
    
    for(i = 0; i < Q; i++){ // initialize our row buffer
      ((unsigned int *)mx->dp)[(i * 32) + myoffset] = 0x80808080;
    }
 
    for(i = 0; i < L; i++){
      rbv = ((unsigned int *)om->rbv[dsq[i]]) + myoffset;
      dp = ((unsigned int *)mx->dp) + myoffset;

      int q= 0;
      while (q < Q-3){ //unroll inner loop 4x
        sv = __vaddss4(mpv, *rbv);
        hv = __vmaxs4(hv, sv);
        mpv = *dp;
        *dp = sv;
        dp += 32; // advance to next
        rbv += 32;  // advance to next 
        sv = __vaddss4(mpv, *rbv);
        hv = __vmaxs4(hv, sv);
        mpv = *dp;
        *dp = sv;
        dp += 32; // advance to next
        rbv += 32;  // advance to next 
        sv = __vaddss4(mpv, *rbv);
        hv = __vmaxs4(hv, sv);
        mpv = *dp;
        *dp = sv;
        dp += 32; // advance to next
        rbv += 32;  // advance to next 
        sv = __vaddss4(mpv, *rbv);
        hv = __vmaxs4(hv, sv);
        mpv = *dp;
        *dp = sv;
        dp += 32; // advance to next
        rbv += 32;  // advance to next 
        q+=4;
      }
      for(; q < Q; q++){ // postamble to finish up
        sv = __vaddss4(mpv, *rbv);
        hv = __vmaxs4(hv, sv);
        mpv = *dp;
        *dp = sv;
        dp += 32; // advance to next
        rbv += 32;  // advance to next 
      }

      // Now, leftshift (memory order) the sv vector to get the next mpv
      sv_shuffle = __shfl_up(sv, 1);  // Get the next core up's value of SV
      if(myoffset == 0){
        sv_shuffle = neginfmask; // special-case the high core in the warp, since it has nobody to grab a value from
      }

      mpv = __byte_perm(sv, sv_shuffle, 0x2107); //left-shifts sv by one byte, puts the high byte of sv_shuffle in the low byte of sv
    }
  }


  unsigned int partner_hv;

  // Done with main loop.  Now reduce answer vector (hv) to one byte for return
  // Reduce 32 4x8-bit quantities to 16
  partner_hv = __shfl_down(hv, 16); 
  if(myoffset < 16){ // only bottom half of the cores continue from here
  

    hv = __vmaxs4(hv, partner_hv);

    // Reduce 6 4x8-bit quantities to 8
    partner_hv = __shfl_down(hv, 8); 
    if(myoffset < 8){ // only bottom half of the cores continue from here
  

      hv = __vmaxs4(hv, partner_hv);

      // Reduce 8 4x8-bit quantities to 4
      partner_hv = __shfl_down(hv, 4); 
      if(myoffset < 4){ // only bottom half of the cores continue from here

        hv = __vmaxs4(hv, partner_hv);

        // Reduce 4 4x8-bit quantities to 2
        partner_hv = __shfl_down(hv, 2); 
        if(myoffset < 2){ // only bottom half of the cores continue from here

          hv = __vmaxs4(hv, partner_hv);
          // Reduce 2 4x8-bit quantities to 1

          partner_hv = __shfl_down(hv, 1); 
          if(myoffset < 1){ // only bottom half of the cores continue from here

            hv = __vmaxs4(hv, partner_hv);

            // now, reduce the final 32 bit quantity to one 8-bit quantity.

            unsigned int temp;

            temp = hv >> 16;

            hv = __vmaxs4(hv, temp);

            temp = hv >> 8;

            hv = __vmaxs4(hv, temp);
            if((myblock == 0) &&(mywarp ==0) && (myoffset == 0)){ // only one thread writes result
              *retval = hv & 255; // low 8 bits of the word is the final result
            }
          }
        }
      }
    }
  }
  } 
  return;
}


// GPU kernel that copies values from the CPU version of an OPROFILE to one on the GPU.  Should generally only be called on one GPU core
__global__ void copy_oprofile_values_to_card(P7_OPROFILE *the_profile, float tauBM, float scale_b, float scale_w, int16_t base_w, int16_t ddbound_w, int L, int M, int V, int max_length, int allocM, int allocQb, int allocQw, int allocQf, int mode, float nj, int is_shadow, int8_t **rbv){

  the_profile->tauBM = tauBM;
  the_profile->scale_b = scale_b;
  the_profile->scale_w = scale_w;
  the_profile->base_w = base_w;
  the_profile->ddbound_w = ddbound_w;
  the_profile->L = L;
  the_profile->M = M;
  the_profile->V = V;
  the_profile->max_length = max_length;
  the_profile->allocM = allocM;
  the_profile->allocQb = allocQb;
  the_profile->allocQw = allocQw;
  the_profile->allocQf = allocQf;
  the_profile->mode = mode;
  the_profile->nj = nj;
  the_profile->is_shadow = is_shadow;
  the_profile->rbv = rbv;
}


// GPU kernel that initializes a filtermx structure
__global__ void initialize_filtermx_on_card(P7_FILTERMX *the_filtermx){
  the_filtermx->M = 0;
  the_filtermx->Vw = 64; // 32 cores * 32 bits = 1024 bits = 128 bytes = 64 * 16 bits
  the_filtermx->allocM = 0;
  the_filtermx->dp = NULL;
  the_filtermx->type = p7F_SSVFILTER;
}


// allocates and populates a P7_OPROFILE structure on a CUDA card that matches the one passed as its argument
P7_OPROFILE *create_oprofile_on_card(P7_OPROFILE *the_profile){
  P7_OPROFILE *cuda_OPROFILE;

  int Q = P7_Q(the_profile->M, the_profile->V);

  if(hipMalloc(&cuda_OPROFILE, sizeof(P7_OPROFILE)) != hipSuccess){
    p7_Fail((char *) "Unable to allocate memory in create_oprofile_on_card");
  }

  // allocate and copy over rbv 2-D array
  unsigned int **cuda_rbv;
  if(hipMalloc(&cuda_rbv, the_profile->abc->Kp * sizeof(unsigned int *)) != hipSuccess){
    p7_Fail((char *) "Unable to allocate memory in create_oprofile_on_card");
  }
  int i;
  unsigned int **cuda_rbv_temp = cuda_rbv; // use this variable to copy rbv pointers into CUDA array 
  for(i = 0; i < the_profile->abc->Kp; i++){
    int *cuda_rbv_entry, *restriped_rbv;
    int restriped_rbv_size;
    restriped_rbv = restripe_char_to_int((char *)(the_profile->rbv[i]), the_profile->V, 32, Q * the_profile->V, &restriped_rbv_size);

    if(hipMalloc(&cuda_rbv_entry, restriped_rbv_size) != hipSuccess){
      p7_Fail((char *) "Unable to allocate memory in create_oprofile_on_card");
    }

    if(hipMemcpy(cuda_rbv_entry, restriped_rbv, restriped_rbv_size, hipMemcpyHostToDevice) != hipSuccess){
      p7_Fail((char *) "Unable to copy data in create_oprofile_on_card");
    }

    if(hipMemcpy(cuda_rbv_temp, &cuda_rbv_entry, sizeof(int *) , hipMemcpyHostToDevice) != hipSuccess){
      p7_Fail((char *) "Unable to copy data in create_oprofile_on_card");
    }
    cuda_rbv_temp +=1;
  }
 
  // copy over base parameters.  Only call this kernel on one core because it just assigns values to fields in the data structure and has no parallelism
  copy_oprofile_values_to_card<<<1,1>>>(cuda_OPROFILE, the_profile->tauBM, the_profile->scale_b, the_profile->scale_w, the_profile->base_w, the_profile->ddbound_w, the_profile->L, the_profile->M, the_profile->V, the_profile->max_length, the_profile->allocM, the_profile->allocQb, the_profile->allocQw, the_profile->allocQf, the_profile->mode, the_profile->nj, the_profile->is_shadow, (int8_t **) cuda_rbv);

 return cuda_OPROFILE;
}

P7_FILTERMX *create_filtermx_on_card(){
  P7_FILTERMX *the_filtermx;
  
  if(hipMalloc(&the_filtermx, sizeof(P7_FILTERMX)) != hipSuccess){
    p7_Fail((char *) "Unable to allocate memory in create_filtermx_on_card");
  }
  initialize_filtermx_on_card<<<1,1>>>(the_filtermx);
  return the_filtermx;
}


char * restripe_char(char *source, int source_chars_per_vector, int dest_chars_per_vector, int source_length, int *dest_length){
  char *dest;
  int dest_num_vectors;
  int source_num_vectors;
  source_num_vectors = source_length/source_chars_per_vector;
  if(source_num_vectors * source_chars_per_vector != source_length){
    source_num_vectors++;
  }
  dest_num_vectors = source_length/dest_chars_per_vector;
  if(dest_num_vectors * dest_chars_per_vector != source_length){
    dest_num_vectors++;  //round up if source length isn't a multiple of the dest vector length
  }

  dest = (char *) malloc(dest_num_vectors * dest_chars_per_vector);
  *dest_length = dest_num_vectors * dest_chars_per_vector;

  int source_pos, dest_pos;
  int i;

  for(i = 0; i < source_length; i++){
    source_pos = ((i % source_num_vectors) * source_chars_per_vector) + (i / source_num_vectors);
    dest_pos = ((i % dest_num_vectors) * dest_chars_per_vector) + (i / dest_num_vectors);
    dest[dest_pos] = source[source_pos];
  }

  // pad out the dest vector with zeroes if necessary
  for(i = source_length; i < *dest_length; i++){
      dest_pos = ((i % dest_num_vectors) * dest_chars_per_vector) + (i / dest_num_vectors);
    dest[dest_pos] = 0;
  }

  return dest;

}


int *restripe_char_to_int(char *source, int source_chars_per_vector, int dest_ints_per_vector, int source_length, int *dest_length){
  int *dest;
  int dest_num_vectors, source_num_vectors, unpadded_dest_vectors;

  source_num_vectors = source_length/source_chars_per_vector;
  if(source_num_vectors * source_chars_per_vector != source_length){
    source_num_vectors++;
  }
  unpadded_dest_vectors = source_length/dest_ints_per_vector;
  if(unpadded_dest_vectors * dest_ints_per_vector != source_length){
    unpadded_dest_vectors++;  //round up if source length isn't a multiple of the dest vector length
  }
 // printf("Unpadded_dest_vectors = %d. ", unpadded_dest_vectors);
  dest_num_vectors = unpadded_dest_vectors + MAX_BAND_WIDTH -1; // add extra vectors for SSV wrap-around
  dest = (int *) malloc(dest_num_vectors * dest_ints_per_vector * sizeof(int));
  *dest_length = dest_num_vectors * dest_ints_per_vector *sizeof(int);
  //printf("Padded dest_num_vectors = %d. Dest_length = %d\n", dest_num_vectors, *dest_length);

  int source_pos, dest_pos;
  int i;

  for(i = 0; i < source_length; i++){
    source_pos = ((i % source_num_vectors) * source_chars_per_vector) + (i / source_num_vectors);
    dest_pos = ((i % unpadded_dest_vectors) * dest_ints_per_vector) + (i / unpadded_dest_vectors);
    dest[dest_pos] = (int) source[source_pos];
  }

  // pad out the dest vector with zeroes if necessary
  for(; i < unpadded_dest_vectors * dest_ints_per_vector; i++){
      dest_pos = ((i % unpadded_dest_vectors) * dest_ints_per_vector) + (i / unpadded_dest_vectors);
    //  printf("Padding 0 at location %d \n", dest_pos);
    dest[dest_pos] = 0;
  }

  // add the extra copies of the early vectors to support the SSV wrap-around
  for(int source = 0; i < dest_num_vectors * dest_ints_per_vector; i++){
    dest[i] = dest[source];
   // printf("Padding from location %d to location %d\n", source, i);
    source++;
  }

  return dest;
}

int
p7_SSVFilter_shell_sse(const ESL_DSQ *dsq, int L, const P7_OPROFILE *om, P7_FILTERMX *fx, float *ret_sc, P7_OPROFILE *card_OPROFILE, P7_FILTERMX *card_FILTERMX)
{
  int      Q          = P7_Q(om->M, p7_VWIDTH_SSE);
  __m128i  hv         = _mm_set1_epi8(-128);
  __m128i  neginfmask = _mm_insert_epi8( _mm_setzero_si128(), -128, 0);
  __m128i *dp;
  __m128i *rbv;
  __m128i  mpv;
  __m128i  sv;
  int8_t   h, *card_h;
  int      i,q;
  int      status;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds, seconds, gcups;
  //char *card_rbv= NULL;
  char *card_dsq;
  int card_Q, warps_per_block;
  dim3 threads_per_block, num_blocks;
  hipError_t err;
  if (( status = p7_filtermx_Reinit(fx, om->M) ) != eslOK) goto FAILURE;
  fx->M    = om->M;
  fx->Vw   = p7_VWIDTH_SSE / sizeof(int16_t); // A hack. FILTERMX wants Vw in units of int16_t. 
  fx->type = p7F_SSVFILTER;
  dp       = (__m128i *) fx->dp;
  card_Q = ((((om->M)-1) / (32)) + 1);
  hipMalloc((void **) &card_h, 1);
  err = hipGetLastError();
  hipMalloc((void**)  &card_dsq, L+8);  //Pad out so that we can grab dsq four bytes at a time
  hipMemcpy(card_dsq, (dsq+ 1), L, hipMemcpyHostToDevice);

 // SSV_start_cuda<<<1, 32>>>(card_FILTERMX, (unsigned int *) card_hv, (unsigned int *) card_mpv, om->M);
  hipEventRecord(start);
  num_blocks.x = 1;
  num_blocks.y = 1;
  num_blocks.z = 1;
  warps_per_block = 1;
  threads_per_block.x = 32;
  threads_per_block.y = warps_per_block;
  threads_per_block.z = 1;

  int **check_array, **check_array_cuda;
  check_array = (int **) malloc(L * sizeof(int *));
  hipMalloc((void **) &check_array_cuda, (L * sizeof(int *)));
  for(int temp = 0; temp < L; temp++){
    int *row_array;
    hipMalloc((void **) &row_array, (om->M * sizeof(int)));
    check_array[temp] = row_array;
  }
  hipMemcpy(check_array_cuda, check_array, (L* sizeof(int)), hipMemcpyHostToDevice);

  SSV_cuda <<<num_blocks, threads_per_block>>>(card_dsq, card_FILTERMX, L, card_Q, card_OPROFILE, card_h, check_array_cuda);
  int8_t h_compare;
  hipMemcpy(&h_compare, card_h, 1, hipMemcpyDeviceToHost);
  hipEventRecord(stop);

  hipEventSynchronize(stop);
  milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  seconds = milliseconds/1000;
  gcups = ((((float) (om->M * L) *(float) NUM_REPS)/seconds)/1e9) * (float)(num_blocks.x * num_blocks.y *num_blocks.z) * (float)warps_per_block;
  printf("M = %d, L = %d, seconds = %f, GCUPS = %f\n", om->M, L, seconds, gcups); 
  

  err = hipGetLastError();
  if(err != hipSuccess){
    printf("Error: %s\n", hipGetErrorString(err));
  }
 
  mpv = hv;
  for (q = 0; q < Q; q++)
    dp[q] = hv;

  for (i = 1; i <= L; i++)
    {
      rbv = (__m128i *) om->rbv[dsq[i]];

      
      for (q = 0; q < Q; q++)
        {
          sv    = _mm_adds_epi8(mpv, rbv[q]);
          hv    = _mm_max_epi8(hv, sv);
          mpv   = dp[q];
          dp[q] = sv;
        }  
      mpv = esl_sse_rightshift_int8(sv, neginfmask);

      // Compare CUDA result and SSE
 /*     hipMemcpy(restriped_rbv, card_dp, card_rbv_length, hipMemcpyDeviceToHost);
      for(j = 0; j < om->M; j++){
          int cuda_index = (j/card_Q) + ((j %card_Q) * 128);
          int cpu_index = (j/Q) + ((j %Q) * om->V);
          char *cuda_dp_char = (char *)restriped_rbv;
          char *cpu_dp_char = (char *) dp;
          char cuda_value = cuda_dp_char[cuda_index];
          char cpu_value = cpu_dp_char[cpu_index];
          if(cpu_value != cuda_value){
            printf("SSV dp miss-match at row %d, position %d: %d (CUDA) vs %d (CPU), indices were %d (CUDA), %d (CPU)\n", i, j, cuda_value, cpu_value, cuda_index, cpu_index);
          }
      }
*/
    //  free(restriped_rbv);
    }
  h = esl_sse_hmax_epi8(hv);
  hipFree(card_h);
  hipFree(card_dsq);


  if(h != h_compare){
    printf("Final result miss-match: %d (CUDA) vs %d (CPU)\n", h_compare, h);
  }

  if (h == 127)  
    { *ret_sc = eslINFINITY; return eslERANGE; }
  else if (h > -128)
    { 
      *ret_sc = ((float) h + 128.) / om->scale_b + om->tauBM - 2.0;   // 2.0 is the tauNN/tauCC "2 nat approximation"
      *ret_sc += 2.0 * logf(2.0 / (float) (L + 2));                   
      return eslOK;
    }
  else 
    {
      *ret_sc = -eslINFINITY;
      return eslOK;
    }
    
 FAILURE:
  *ret_sc = -eslINFINITY;
  return status;
}


static ESL_OPTIONS options[] = {
  /* name           type      default  env  range  toggles reqs incomp  help                               docgroup*/
  { (char *) "-h",        eslARG_NONE,  FALSE,  NULL, NULL,   NULL,  NULL, NULL,  (char *) "show brief help on version and usage",  0 },
  {  (char *) "-s",        eslARG_INT,      (char *) "0",  NULL, NULL,   NULL,  NULL, NULL,  (char *) "set random number seed to <n>",         0 },
  {  0, 0, 0, 0, 0, 0, 0, 0, 0, 0 },
};
static char usage[]  = "[-options] <hmmfile> <seqfile>";
static char banner[] = "px, the first parallel tests of H4";

int
main(int argc, char **argv)
{
  ESL_GETOPTS    *go      = p7_CreateDefaultApp(options, 2, argc, argv, banner, usage);
  char           *hmmfile = esl_opt_GetArg(go, 1);
  char           *seqfile = esl_opt_GetArg(go, 2);
  ESL_ALPHABET   *abc     = NULL;
  P7_HMMFILE     *hfp     = NULL;
  P7_BG          *bg      = NULL;
  P7_HMM         *hmm     = NULL;
  P7_PROFILE     *gm      = NULL;
  P7_OPROFILE    *om      = NULL;
  ESL_DSQDATA    *dd      = NULL;
  P7_ENGINE      *eng     = NULL;
  ESL_DSQDATA_CHUNK *chu = NULL;
  int             ncore   = 1;
  int  i;
  int             status;

  /* Read in one HMM */
  if (p7_hmmfile_OpenE(hmmfile, NULL, &hfp, NULL) != eslOK) p7_Fail( (char *) "Failed to open HMM file %s", hmmfile);
  if (p7_hmmfile_Read(hfp, &abc, &hmm)            != eslOK) p7_Fail( (char *) "Failed to read HMM");
  
  /* Configure a profile from the HMM */
  bg = p7_bg_Create(abc);
  gm = p7_profile_Create (hmm->M, abc);
  om = p7_oprofile_Create(hmm->M, abc);
  p7_profile_Config   (gm, hmm, bg);
  p7_oprofile_Convert (gm, om);
  P7_OPROFILE *card_OPROFILE;
  card_OPROFILE = create_oprofile_on_card((P7_OPROFILE *) om);
  P7_FILTERMX *card_FILTERMX;
  card_FILTERMX = create_filtermx_on_card();
  p7_bg_SetFilter(bg, om->M, om->compo);

  //uint64_t sequence_id = 0;
  uint64_t num_hits = 0;
  int count;
  hipGetDeviceCount(&count);
  printf("Found %d CUDA devices\n", count);
  /* Open sequence database */
  status = esl_dsqdata_Open(&abc, seqfile, ncore, &dd);
  if      (status == eslENOTFOUND) p7_Fail( (char *) "Failed to open dsqdata files:\n  %s",    dd->errbuf);
  else if (status == eslEFORMAT)   p7_Fail( (char *) "Format problem in dsqdata files:\n  %s", dd->errbuf);
  else if (status != eslOK)        p7_Fail( (char *) "Unexpected error in opening dsqdata (code %d)", status);

  eng = p7_engine_Create(abc, NULL, NULL, gm->M, 400);

  while (( status = esl_dsqdata_Read(dd, &chu)) == eslOK && num_hits < 5)  
    {
      for (i = 0; i < 5 /* chu->N */; i++)
	{
	  p7_bg_SetLength(bg, (int) chu->L[i]);            // TODO: remove need for cast
	  p7_oprofile_ReconfigLength(om, (int) chu->L[i]); //         (ditto)
	  
	  //	  printf("seq %d %s\n", chu->i0+i, chu->name[i]);
    float ssv_score;

    p7_SSVFilter_shell_sse(chu->dsq[i], chu->L[i], om, eng->fx ,&ssv_score, card_OPROFILE, card_FILTERMX);
	 

	  p7_engine_Reuse(eng);
    num_hits++;
	}
      esl_dsqdata_Recycle(dd, chu);
    }
    printf("Saw %ld sequences\n", num_hits);
  /*esl_dsqdata_Close(dd);
  p7_oprofile_Destroy(om);
  p7_profile_Destroy(gm);
  p7_hmm_Destroy(hmm);
  p7_bg_Destroy(bg);
  p7_hmmfile_Close(hfp);
  esl_alphabet_Destroy(abc);
  esl_getopts_Destroy(go); */
  exit(0);
}





